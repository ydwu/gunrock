// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file sssp_app.cu
 *
 * @brief single-source shortest path (SSSP) application
 */

#include <gunrock/gunrock.h>

// graph construction utilities
#include <gunrock/graphio/market.cuh>

// single-source shortest path includes
#include <gunrock/app/sssp/sssp_enactor.cuh>
#include <gunrock/app/sssp/sssp_problem.cuh>
#include <gunrock/app/sssp/sssp_functor.cuh>

#include <moderngpu.cuh>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::sssp;

/**
 * @brief Test_Parameter structure
 */
struct Test_Parameter : gunrock::app::TestParameter_Base
{
public:
    bool   mark_predecessors;
    int    delta_factor;
    double max_queue_sizing1;

    Test_Parameter()
    {
        delta_factor      =    32;
        mark_predecessors = false;
        max_queue_sizing1 =  -1.0;
    }

    ~Test_Parameter()
    {
    }
};

template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT,
    bool DEBUG,
    bool SIZE_CHECK,
    bool MARK_PREDECESSORS >
void runSSSP(GRGraph* output, Test_Parameter *parameter);

/**
 * @brief Run test
 *
 * @tparam VertexId   Vertex identifier type
 * @tparam Value      Attribute type
 * @tparam SizeT      Graph size type
 * @tparam INSTRUMENT Keep kernels statics
 * @tparam DEBUG      Keep debug statics
 * @tparam SIZE_CHECK Enable size check
 *
 * @praam[out] output    Pointer to output graph structure of the problem
 * @param[in]  parameter primitive-specific test parameters
 */
template <
    typename    VertexId,
    typename    Value,
    typename    SizeT,
    bool        INSTRUMENT,
    bool        DEBUG,
    bool        SIZE_CHECK >
void markPredecessorsSSSP(GRGraph* output, Test_Parameter *parameter)
{
    if (parameter->mark_predecessors)
        runSSSP<VertexId, Value, SizeT, INSTRUMENT,
                DEBUG, SIZE_CHECK,  true>(output, parameter);
    else
        runSSSP<VertexId, Value, SizeT, INSTRUMENT,
                DEBUG, SIZE_CHECK, false>(output, parameter);
}

/**
 * @brief Run test
 *
 * @tparam VertexId   Vertex identifier type
 * @tparam Value      Attribute type
 * @tparam SizeT      Graph size type
 * @tparam INSTRUMENT Keep kernels statics
 * @tparam DEBUG      Keep debug statics
 *
 * @praam[out] output    Pointer to output graph structure of the problem
 * @param[in]  parameter primitive-specific test parameters
 */
template <
    typename      VertexId,
    typename      Value,
    typename      SizeT,
    bool          INSTRUMENT,
    bool          DEBUG >
void sizeCheckSSSP(GRGraph* output, Test_Parameter *parameter)
{
    if (parameter->size_check)
        markPredecessorsSSSP<VertexId, Value, SizeT, INSTRUMENT,
                             DEBUG,  true>(output, parameter);
    else
        markPredecessorsSSSP<VertexId, Value, SizeT, INSTRUMENT,
                             DEBUG, false>(output, parameter);
}

/**
 * @brief Run test
 *
 * @tparam VertexId   Vertex identifier type
 * @tparam Value      Attribute type
 * @tparam SizeT      Graph size type
 * @tparam INSTRUMENT Keep kernels statics
 *
 * @praam[out] output    Pointer to output graph structure of the problem
 * @param[in]  parameter primitive-specific test parameters
 */
template <
    typename    VertexId,
    typename    Value,
    typename    SizeT,
    bool        INSTRUMENT >
void debugSSSP(GRGraph* output, Test_Parameter *parameter)
{
    if (parameter->debug)
        sizeCheckSSSP<VertexId, Value, SizeT, INSTRUMENT,
                      true>(output, parameter);
    else
        sizeCheckSSSP<VertexId, Value, SizeT, INSTRUMENT,
                      false>(output, parameter);
}

/**
 * @brief Run test
 *
 * @tparam VertexId Vertex identifier type
 * @tparam Value    Attribute type
 * @tparam SizeT    Graph size type
 *
 * @praam[out] output    Pointer to output graph structure of the problem
 * @param[in]  parameter primitive-specific test parameters
 */
template <
    typename      VertexId,
    typename      Value,
    typename      SizeT >
void instrumentedSSSP(GRGraph* output, Test_Parameter *parameter)
{
    if (parameter->instrumented)
        debugSSSP<VertexId, Value, SizeT,  true>(output, parameter);
    else
        debugSSSP<VertexId, Value, SizeT, false>(output, parameter);
}

/**
 * @brief Run test
 *
 * @tparam VertexId          Vertex identifier type
 * @tparam Value             Attribute type
 * @tparam SizeT             Graph size type
 * @tparam INSTRUMENT        Keep kernels statics
 * @tparam DEBUG             Keep debug statics
 * @tparam SIZE_CHECK        Enable size check
 * @tparam MARK_PREDECESSORS Enable mark predecessors
 *
 * @praam[out] output    Pointer to output graph structure of the problem
 * @param[in]  parameter primitive-specific test parameters
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT,
    bool DEBUG,
    bool SIZE_CHECK,
    bool MARK_PREDECESSORS >
void runSSSP(GRGraph* output, Test_Parameter *parameter)
{
    typedef SSSPProblem < VertexId,
            SizeT,
            Value,
            MARK_PREDECESSORS > SsspProblem;

    typedef SSSPEnactor < SsspProblem,
            INSTRUMENT,
            DEBUG,
            SIZE_CHECK > SsspEnactor;

    Csr<VertexId, Value, SizeT>
    *graph = (Csr<VertexId, Value, SizeT>*)parameter->graph;
    bool          quiet              = parameter -> g_quiet;
    VertexId      src                = (VertexId)parameter -> src;
    int           max_grid_size      = parameter -> max_grid_size;
    int           num_gpus           = parameter -> num_gpus;
    double        max_queue_sizing   = parameter -> max_queue_sizing;
    double        max_in_sizing      = parameter -> max_in_sizing;
    ContextPtr   *context            = (ContextPtr*)parameter -> context;
    std::string   partition_method   = parameter -> partition_method;
    int          *gpu_idx            = parameter -> gpu_idx;
    hipStream_t *streams            = parameter -> streams;
    float         partition_factor   = parameter -> partition_factor;
    int           partition_seed     = parameter -> partition_seed;
    bool          g_stream_from_host = parameter -> g_stream_from_host;
    int           delta_factor       = parameter -> delta_factor;
    int           traversal_mode     = parameter -> traversal_mode;
    size_t       *org_size           = new size_t[num_gpus];
    // Allocate host-side label arrays
    Value    *h_labels = new Value[graph->nodes];
    VertexId *h_preds  = MARK_PREDECESSORS ? new VertexId[graph->nodes] : NULL;

    for (int gpu = 0; gpu < num_gpus; gpu++)
    {
        size_t dummy;
        hipSetDevice(gpu_idx[gpu]);
        hipMemGetInfo(&(org_size[gpu]), &dummy);
    }

    SsspEnactor* enactor = new SsspEnactor(num_gpus, gpu_idx);  // enactor map
    SsspProblem* problem = new SsspProblem;  // Allocate problem on GPU

    util::GRError(
        problem->Init(
            g_stream_from_host,
            graph,
            NULL,
            num_gpus,
            gpu_idx,
            partition_method,
            streams,
            delta_factor,
            max_queue_sizing,
            max_in_sizing,
            partition_factor,
            partition_seed),
        "Problem SSSP Initialization Failed", __FILE__, __LINE__);

    util::GRError(
        enactor->Init (context, problem, max_grid_size, traversal_mode),
        "SSSP Enactor init failed", __FILE__, __LINE__);

    // Perform SSSP
    CpuTimer cpu_timer;

    util::GRError(
        problem->Reset(src, enactor->GetFrontierType(), max_queue_sizing),
        "SSSP Problem Data Reset Failed", __FILE__, __LINE__);
    util::GRError(
        enactor->Reset(), "SSSP Enactor Reset failed", __FILE__, __LINE__);

    if (!quiet) { printf("__________________________\n"); fflush(stdout); }
    cpu_timer.Start();
    util::GRError(
        enactor->Enact(src, traversal_mode),
        "SSSP Problem Enact Failed", __FILE__, __LINE__);
    cpu_timer.Stop();
    if (!quiet) { printf("--------------------------\n"); fflush(stdout); }
    float elapsed = cpu_timer.ElapsedMillis();

    // Copy out results
    util::GRError(
        problem->Extract(h_labels, h_preds),
        "SSSP Problem Data Extraction Failed", __FILE__, __LINE__);

    output->node_value1 = (Value*)&h_labels[0];
    if (MARK_PREDECESSORS) output->node_value2 = (VertexId*)&h_preds[0];

    if (!quiet)
    {
        printf(" GPU Single-Source Shortest Path finished in %lf msec.\n", elapsed);
    }

    // Clean up
    if (org_size) { delete[] org_size; org_size = NULL; }
    if (enactor ) { delete   enactor ; enactor  = NULL; }
    if (problem ) { delete   problem ; problem  = NULL; }
}

/**
 * @brief Dispatch function to handle configurations
 *
 * @param[out] grapho  Pointer to output graph structure of the problem
 * @param[in]  graphi  Pointer to input graph we need to process on
 * @param[in]  config  Primitive-specific configurations
 * @param[in]  data_t  Data type configurations
 * @param[in]  context ModernGPU context
 * @param[in]  streams CUDA stream
 */
void dispatchSSSP(
    GRGraph*       grapho,
    const GRGraph* graphi,
    const GRSetup  config,
    const GRTypes  data_t,
    ContextPtr*    context,
    hipStream_t*  streams)
{
    Test_Parameter *parameter = new Test_Parameter;
    parameter->context  = context;
    parameter->streams  = streams;
    parameter->g_quiet  = config.quiet;
    parameter->num_gpus = config.num_devices;
    parameter->gpu_idx  = config.device_list;
    parameter->delta_factor = config.delta_factor;
    parameter->traversal_mode = config.traversal_mode;
    parameter->mark_predecessors  = config.mark_predecessors;

    switch (data_t.VTXID_TYPE)
    {
    case VTXID_INT:
    {
        switch (data_t.SIZET_TYPE)
        {
        case SIZET_INT:
        {
            switch (data_t.VALUE_TYPE)
            {
            case VALUE_INT:    // template type = <int, int, int>
            {
                Csr<int, int, int> csr(false);
                csr.nodes = graphi->num_nodes;
                csr.edges = graphi->num_edges;
                csr.row_offsets    = (int*)graphi->row_offsets;
                csr.column_indices = (int*)graphi->col_indices;
                csr.edge_values    = (int*)graphi->edge_values;
                parameter->graph = &csr;

                // determine source vertex to start
                switch (config.source_mode)
                {
                case randomize:
                {
                    parameter->src = graphio::RandomNode(csr.nodes);
                    break;
                }
                case largest_degree:
                {
                    int max_deg = 0;
                    parameter->src = csr.GetNodeWithHighestDegree(max_deg);
                    break;
                }
                case manually:
                {
                    parameter->src = config.source_vertex;
                    break;
                }
                default:
                {
                    parameter->src = 0;
                    break;
                }
                }
                if (!parameter->g_quiet)
                {
                    printf(" source: %lld\n", (long long) parameter->src);
                }
                
                instrumentedSSSP<int, int, int>(grapho, parameter);

                // reset for free memory
                csr.row_offsets    = NULL;
                csr.column_indices = NULL;
                csr.edge_values    = NULL;
                break;
            }
            case VALUE_UINT:    // template type = <int, uint, int>
            {
                // not support yet
                printf("Not Yet Support This DataType Combination.\n");
                break;
            }
            case VALUE_FLOAT:
            {
                // template type = <int, float, int>
                // not support yet
                printf("Not Yet Support This DataType Combination.\n");
                break;
            }
            }
            break;
        }
        }
        break;
    }
    }
}

/*
 * @brief Entry of gunrock_sssp function
 *
 * @param[out] grapho Pointer to output graph structure of the problem
 * @param[in]  graphi Pointer to input graph we need to process on
 * @param[in]  config Gunrock primitive specific configurations
 * @param[in]  data_t Gunrock data type structure
 */
void gunrock_sssp(
    GRGraph*       grapho,
    const GRGraph* graphi,
    const GRSetup  config,
    const GRTypes  data_t)
{
    // GPU-related configurations
    int           num_gpus =    0;
    int           *gpu_idx = NULL;
    ContextPtr    *context = NULL;
    hipStream_t  *streams = NULL;

    num_gpus = config.num_devices;
    gpu_idx  = new int [num_gpus];
    for (int i = 0; i < num_gpus; ++i)
    {
        gpu_idx[i] = config.device_list[i];
    }

    // Create streams and MordernGPU context for each GPU
    streams = new hipStream_t[num_gpus * num_gpus * 2];
    context = new ContextPtr[num_gpus * num_gpus];
    if (!config.quiet) { printf(" using %d GPUs:", num_gpus); }
    for (int gpu = 0; gpu < num_gpus; ++gpu)
    {
        if (!config.quiet) { printf(" %d ", gpu_idx[gpu]); }
        util::SetDevice(gpu_idx[gpu]);
        for (int i = 0; i < num_gpus * 2; ++i)
        {
            int _i = gpu * num_gpus * 2 + i;
            util::GRError(hipStreamCreate(&streams[_i]),
                          "hipStreamCreate fialed.", __FILE__, __LINE__);
            if (i < num_gpus)
            {
                context[gpu * num_gpus + i] =
                    mgpu::CreateCudaDeviceAttachStream(gpu_idx[gpu],
                                                       streams[_i]);
            }
        }
    }
    if (!config.quiet) { printf("\n"); }

    dispatchSSSP(grapho, graphi, config, data_t, context, streams);
}

/*
 * @brief Simple interface take in CSR arrays as input
 *
 * @param[out] distances   Return shortest distance to source per nodes
 * @param[in]  num_nodes   Number of nodes of the input graph
 * @param[in]  num_edges   Number of edges of the input graph
 * @param[in]  row_offsets CSR-formatted graph input row offsets
 * @param[in]  col_indices CSR-formatted graph input column indices
 * @param[in]  source      Source to begin traverse
 */
void sssp(
    unsigned int*       distances,
    const int           num_nodes,
    const int           num_edges,
    const int*          row_offsets,
    const int*          col_indices,
    const unsigned int* edge_values,
    const int           source)
{
    struct GRTypes data_t;          // primitive-specific data types
    data_t.VTXID_TYPE = VTXID_INT;  // integer vertex identifier
    data_t.SIZET_TYPE = SIZET_INT;  // integer graph size type
    data_t.VALUE_TYPE = VALUE_INT;  // integer attributes type

    struct GRSetup config;                // primitive-specific configures
    int list[] = {0, 1, 2, 3};            // device to run algorithm
    config.num_devices = sizeof(list) / sizeof(list[0]);  // number of devices
    config.device_list       = list;      // device list to run algorithm
    config.source_mode       = manually;  // manually setting source vertex
    config.source_vertex     = source;    // source vertex to start
    config.delta_factor      =    32;     // delta factor for SSSP
    config.mark_predecessors = false;     // do not mark predecessors
    config.traversal_mode    =     0;     // 0 for Load balanced partition
    config.max_queue_sizing  =  1.0f;     // maximum queue sizing factor

    struct GRGraph *grapho = (struct GRGraph*)malloc(sizeof(struct GRGraph));
    struct GRGraph *graphi = (struct GRGraph*)malloc(sizeof(struct GRGraph));

    graphi->num_nodes   = num_nodes;  // setting graph nodes
    graphi->num_edges   = num_edges;  // setting graph edges
    graphi->row_offsets = (void*)&row_offsets[0];  // setting row_offsets
    graphi->col_indices = (void*)&col_indices[0];  // setting col_indices
    graphi->edge_values = (void*)&edge_values[0];  // setting edge_values
    printf(" loaded %d nodes and %d edges\n", num_nodes, num_edges);

    gunrock_sssp(grapho, graphi, config, data_t);
    memcpy(distances, (int*)grapho->node_value1, num_nodes * sizeof(int));

    if (graphi) free(graphi);
    if (grapho) free(grapho);
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:

// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file pr_app.cu
 *
 * @brief Gunrock PageRank application
 */

#include <gunrock/gunrock.h>

// graph construction utilities
#include <gunrock/graphio/market.cuh>

// page-rank includes
#include <gunrock/app/pr/pr_enactor.cuh>
#include <gunrock/app/pr/pr_problem.cuh>
#include <gunrock/app/pr/pr_functor.cuh>

#include <moderngpu.cuh>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::pr;

/**
 * @brief Test_Parameter structure
 */
struct Test_Parameter : gunrock::app::TestParameter_Base
{
public:
    float    delta          ;  // Delta value for PageRank
    float    error          ;  // Error threshold PageRank
    int      max_iter       ;  // Maximum number of iteration

    Test_Parameter()
    {
        delta    = 0.85f;
        error    = 0.01f;
        max_iter =    50;
        src      =    -1;
    }
    ~Test_Parameter()
    {
    }
};

template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT,
    bool DEBUG,
    bool SIZE_CHECK >
void runPageRank(GRGraph *output, Test_Parameter *parameter);

/**
 * @brief Run test
 *
 * @tparam VertexId   Vertex identifier type
 * @tparam Value      Attribute type
 * @tparam SizeT      Graph size type
 * @tparam INSTRUMENT Keep kernels statics
 * @tparam DEBUG      Keep debug statics
 *
 * @praam[out] output    Pointer to output graph structure of the problem
 * @param[in]  parameter primitive-specific test parameters
 */
template <
    typename      VertexId,
    typename      Value,
    typename      SizeT,
    bool          INSTRUMENT,
    bool          DEBUG >
void sizeCheckPageRank(GRGraph *output, Test_Parameter *parameter)
{
    if (parameter->size_check)
        runPageRank<VertexId, Value, SizeT, INSTRUMENT, DEBUG,
                    true > (output, parameter);
    else
        runPageRank<VertexId, Value, SizeT, INSTRUMENT, DEBUG,
                    false> (output, parameter);
}

/**
 * @brief Run test
 *
 * @tparam VertexId   Vertex identifier type
 * @tparam Value      Attribute type
 * @tparam SizeT      Graph size type
 * @tparam INSTRUMENT Keep kernels statics
 *
 * @praam[out] output    Pointer to output graph structure of the problem
 * @param[in]  parameter primitive-specific test parameters
 */
template <
    typename    VertexId,
    typename    Value,
    typename    SizeT,
    bool        INSTRUMENT >
void debugPageRank(GRGraph *output, Test_Parameter *parameter)
{
    if (parameter->debug)
        sizeCheckPageRank<VertexId, Value, SizeT, INSTRUMENT,
                          true > (output, parameter);
    else
        sizeCheckPageRank<VertexId, Value, SizeT, INSTRUMENT,
                          false> (output, parameter);
}

/**
 * @brief Run test
 *
 * @tparam VertexId Vertex identifier type
 * @tparam Value    Attribute type
 * @tparam SizeT    Graph size type
 *
 * @praam[out] output    Pointer to output graph structure of the problem
 * @param[in]  parameter primitive-specific test parameters
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT >
void runPageRank(GRGraph *output, Test_Parameter* parameter)
{
    if (parameter->instrumented)
        debugPageRank<VertexId, Value, SizeT,  true>(output, parameter);
    else
        debugPageRank<VertexId, Value, SizeT, false>(output, parameter);
}

/**
 * @brief Run test
 *
 * @tparam VertexId   Vertex identifier type
 * @tparam Value      Attribute type
 * @tparam SizeT      Graph size type
 * @tparam INSTRUMENT Keep kernels statics
 * @tparam DEBUG      Keep debug statics
 * @tparam SIZE_CHECK Enable size check
 *
 * @praam[out] output    Pointer to output graph structure of the problem
 * @param[in]  parameter primitive-specific test parameters
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT,
    bool DEBUG,
    bool SIZE_CHECK >
void runPageRank(GRGraph *output, Test_Parameter *parameter)
{
    typedef PRProblem < VertexId,
            SizeT,
            Value > PrProblem;

    typedef PREnactor < PrProblem,
            INSTRUMENT,
            DEBUG,
            SIZE_CHECK > PrEnactor;

    Csr<VertexId, Value, SizeT>
    *graph              = (Csr<VertexId, Value, SizeT>*)parameter->graph;
    bool          quiet              = parameter -> g_quiet;
    int           max_grid_size      = parameter -> max_grid_size;
    int           num_gpus           = parameter -> num_gpus;
    double        max_queue_sizing   = parameter -> max_queue_sizing;
    double        max_in_sizing      = parameter -> max_in_sizing;
    ContextPtr   *context            = (ContextPtr*)parameter -> context;
    std::string   partition_method   = parameter -> partition_method;
    int          *gpu_idx            = parameter -> gpu_idx;
    hipStream_t *streams            = parameter -> streams;
    float         partition_factor   = parameter -> partition_factor;
    int           partition_seed     = parameter -> partition_seed;
    bool          g_stream_from_host = parameter -> g_stream_from_host;
    VertexId      src                = parameter -> src;
    Value         delta              = parameter -> delta;
    Value         error              = parameter -> error;
    SizeT         max_iter           = parameter -> max_iter;
    int           traversal_mode     = parameter -> traversal_mode;
    size_t       *org_size           = new size_t  [num_gpus];
    // Allocate host-side label arrays
    Value        *h_rank             = new Value   [graph->nodes];
    VertexId     *h_node_id          = new VertexId[graph->nodes];

    for (int gpu = 0; gpu < num_gpus; gpu++)
    {
        size_t dummy;
        hipSetDevice(gpu_idx[gpu]);
        hipMemGetInfo(&(org_size[gpu]), &dummy);
    }

    PrEnactor* enactor = new PrEnactor(num_gpus, gpu_idx);  // enactor map
    PrProblem *problem = new PrProblem;  // Allocate problem on GPU

    util::GRError(
        problem->Init(
            g_stream_from_host,
            graph,
            NULL,
            num_gpus,
            gpu_idx,
            partition_method,
            streams,
            max_queue_sizing,
            max_in_sizing,
            partition_factor,
            partition_seed),
        "PR Initialization Failed", __FILE__, __LINE__);
    util::GRError(
        enactor->Init(context, problem, traversal_mode, max_grid_size),
        "PR Enactor Init failed", __FILE__, __LINE__);

    // Perform PageRank
    CpuTimer cpu_timer;

    util::GRError(
        problem->Reset(src, delta, error, max_iter,
                       enactor->GetFrontierType(), max_queue_sizing),
        "PR Problem Data Reset Failed", __FILE__, __LINE__);
    util::GRError(
        enactor->Reset(), "PR Enactor Reset Reset failed", __FILE__, __LINE__);

    if (!quiet)
    {
        printf("_________________________________________\n"); fflush(stdout);
    }
    cpu_timer.Start();
    util::GRError(
        enactor->Enact(traversal_mode),
        "PR Problem Enact Failed", __FILE__, __LINE__);
    cpu_timer.Stop();
    if (!quiet)
    {
        printf("-----------------------------------------\n"); fflush(stdout);
    }
    float elapsed = cpu_timer.ElapsedMillis();

    // Copy out results
    util::GRError(
        problem->Extract(h_rank, h_node_id),
        "PR Problem Data Extraction Failed", __FILE__, __LINE__);

    float total_pr = 0;
    for (int i = 0; i < graph->nodes; ++i)
    {
        total_pr += h_rank[i];
    }
    if (!quiet) { printf(" Total rank : %lf\n", total_pr); }

    output->node_value1 = (Value*)&h_rank[0];
    output->node_value2 = (VertexId*)&h_node_id[0];

    if (!quiet) { printf(" GPU PageRank finished in %lf msec.\n", elapsed); }

    // Clean up
    if (org_size) { delete org_size; org_size = NULL; }
    if (problem ) { delete problem ; problem  = NULL; }
    if (enactor ) { delete enactor ; enactor  = NULL; }
}

/**
 * @brief Dispatch function to handle configurations
 *
 * @param[out] grapho  Pointer to output graph structure of the problem
 * @param[in]  graphi  Pointer to input graph we need to process on
 * @param[in]  config  Primitive-specific configurations
 * @param[in]  data_t  Data type configurations
 * @param[in]  context ModernGPU context
 * @param[in]  streams CUDA stream
 */
void dispatchPageRank(
    GRGraph       *grapho,
    const GRGraph *graphi,
    const GRSetup  config,
    const GRTypes  data_t,
    ContextPtr*    context,
    hipStream_t*  streams)
{
    Test_Parameter *parameter = new Test_Parameter;
    parameter->context      =  context;
    parameter->streams      =  streams;
    parameter->g_quiet      = config.quiet;
    parameter->num_gpus     = config.num_devices;
    parameter->gpu_idx      = config.device_list;
    parameter->delta        = config.pagerank_delta;
    parameter->error        = config.pagerank_error;
    parameter->max_iter     = config.max_iters;
    parameter->g_undirected = true;

    switch (data_t.VTXID_TYPE)
    {
    case VTXID_INT:
    {
        switch (data_t.SIZET_TYPE)
        {
        case SIZET_INT:
        {
            switch (data_t.VALUE_TYPE)
            {
            case VALUE_INT:    // template type = <int, int, int>
            {
                printf("Not Yet Support This DataType Combination.\n");
                break;
            }
            case VALUE_UINT:    // template type = <int, uint, int>
            {
                printf("Not Yet Support This DataType Combination.\n");
                break;
            }
            case VALUE_FLOAT:    // template type = <int, float, int>
            {
                // build input csr format graph
                Csr<int, int, int> csr(false);
                csr.nodes = graphi->num_nodes;
                csr.edges = graphi->num_edges;
                csr.row_offsets    = (int*)graphi->row_offsets;
                csr.column_indices = (int*)graphi->col_indices;
                parameter->graph = &csr;

                runPageRank<int, float, int>(grapho, parameter);

                // reset for free memory
                csr.row_offsets    = NULL;
                csr.column_indices = NULL;
                break;
            }
            }
            break;
        }
        }
        break;
    }
    }
}

/*
 * @brief Entry of gunrock_pagerank function
 *
 * @param[out] grapho Pointer to output graph structure of the problem
 * @param[in]  graphi Pointer to input graph we need to process on
 * @param[in]  config Gunrock primitive specific configurations
 * @param[in]  data_t Gunrock data type structure
 */
void gunrock_pagerank(
    GRGraph       *grapho,
    const GRGraph *graphi,
    const GRSetup  config,
    const GRTypes  data_t)
{
    // GPU-related configurations
    int           num_gpus =    0;
    int           *gpu_idx = NULL;
    ContextPtr    *context = NULL;
    hipStream_t  *streams = NULL;

    num_gpus = config.num_devices;
    gpu_idx  = new int [num_gpus];
    for (int i = 0; i < num_gpus; ++i)
    {
        gpu_idx[i] = config.device_list[i];
    }

    // Create streams and MordernGPU context for each GPU
    streams = new hipStream_t[num_gpus * num_gpus * 2];
    context = new ContextPtr[num_gpus * num_gpus];
    if (!config.quiet) { printf(" using %d GPUs:", num_gpus); }
    for (int gpu = 0; gpu < num_gpus; ++gpu)
    {
        if (!config.quiet) { printf(" %d ", gpu_idx[gpu]); }
        util::SetDevice(gpu_idx[gpu]);
        for (int i = 0; i < num_gpus * 2; ++i)
        {
            int _i = gpu * num_gpus * 2 + i;
            util::GRError(hipStreamCreate(&streams[_i]),
                          "hipStreamCreate fialed.", __FILE__, __LINE__);
            if (i < num_gpus)
            {
                context[gpu * num_gpus + i] =
                    mgpu::CreateCudaDeviceAttachStream(gpu_idx[gpu],
                                                       streams[_i]);
            }
        }
    }
    if (!config.quiet) { printf("\n"); }

    dispatchPageRank(grapho, graphi, config, data_t, context, streams);
}

/*
 * @brief Simple interface take in CSR arrays as input
 *
 * @param[out] node_ids    Return top-ranked vertex IDs
 * @param[out] pagerank    Return PageRank scores per node
 * @param[in]  num_nodes   Number of nodes of the input graph
 * @param[in]  num_edges   Number of edges of the input graph
 * @param[in]  row_offsets CSR-formatted graph input row offsets
 * @param[in]  col_indices CSR-formatted graph input column indices
 * @param[in]  source      Source to begin traverse
 */
void pagerank(
    int*                node_ids,
    float*              pagerank,
    const int           num_nodes,
    const int           num_edges,
    const int*          row_offsets,
    const int*          col_indices)
{
    struct GRTypes data_t;            // primitive-specific data types
    data_t.VTXID_TYPE = VTXID_INT;    // integer vertex identifier
    data_t.SIZET_TYPE = SIZET_INT;    // integer graph size type
    data_t.VALUE_TYPE = VALUE_FLOAT;  // float attributes type

    struct GRSetup config;            // primitive-specific configures
    int list[] = {0, 1, 2, 3};        // device to run algorithm
    config.num_devices = sizeof(list) / sizeof(list[0]);  // number of devices
    config.device_list    =  list;    // device list to run algorithm
    config.pagerank_delta = 0.85f;    // default delta value
    config.pagerank_error = 0.01f;    // default error threshold
    config.max_iters      =    50;    // maximum number of iterations
    config.top_nodes      =    10;    // number of top nodes

    struct GRGraph *grapho = (struct GRGraph*)malloc(sizeof(struct GRGraph));
    struct GRGraph *graphi = (struct GRGraph*)malloc(sizeof(struct GRGraph));

    graphi->num_nodes   = num_nodes;  // setting graph nodes
    graphi->num_edges   = num_edges;  // setting graph edges
    graphi->row_offsets = (void*)&row_offsets[0];  // setting row_offsets
    graphi->col_indices = (void*)&col_indices[0];  // setting col_indices
    printf(" loaded %d nodes and %d edges\n", num_nodes, num_edges);

    gunrock_pagerank(grapho, graphi, config, data_t);
    memcpy(pagerank, (float*)grapho->node_value1, num_nodes * sizeof(float));
    memcpy(node_ids, (  int*)grapho->node_value2, num_nodes * sizeof(  int));

    if (graphi) free(graphi);
    if (grapho) free(grapho);
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:

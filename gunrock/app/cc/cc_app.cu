// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file cc_app.cu
 *
 * @brief connected component (CC) application
 */

#include <gunrock/gunrock.h>

// graph construction utilities
#include <gunrock/graphio/market.cuh>

// connected component includes
#include <gunrock/app/cc/cc_enactor.cuh>
#include <gunrock/app/cc/cc_problem.cuh>
#include <gunrock/app/cc/cc_functor.cuh>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::cc;

/**
 * @brief Test_Parameter structure
 */
struct Test_Parameter : gunrock::app::TestParameter_Base
{
public:
    Test_Parameter()  { }
    ~Test_Parameter() { }
};

template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT,
    bool DEBUG,
    bool SIZE_CHECK >
void runCC(GRGraph* output, Test_Parameter *parameter);

/**
 * @brief Run test
 *
 * @tparam VertexId   Vertex identifier type
 * @tparam Value      Attribute type
 * @tparam SizeT      Graph size type
 * @tparam INSTRUMENT Keep kernels statics
 * @tparam DEBUG      Keep debug statics
 *
 * @praam[out] output    Pointer to output graph structure of the problem
 * @param[in]  parameter primitive-specific test parameters
 */
template <
    typename      VertexId,
    typename      Value,
    typename      SizeT,
    bool          INSTRUMENT,
    bool          DEBUG >
void sizeCheckCC(GRGraph* output, Test_Parameter *parameter)
{
    if (parameter->size_check)
        runCC<VertexId, Value, SizeT, INSTRUMENT, DEBUG,
              true > (output, parameter);

    else
        runCC<VertexId, Value, SizeT, INSTRUMENT, DEBUG,
              false> (output, parameter);
}

/**
 * @brief Run test
 *
 * @tparam VertexId   Vertex identifier type
 * @tparam Value      Attribute type
 * @tparam SizeT      Graph size type
 * @tparam INSTRUMENT Keep kernels statics
 *
 * @praam[out] output    Pointer to output graph structure of the problem
 * @param[in]  parameter primitive-specific test parameters
 */
template <
    typename    VertexId,
    typename    Value,
    typename    SizeT,
    bool        INSTRUMENT >
void debugCC(GRGraph* output, Test_Parameter *parameter)
{
    if (parameter->debug)
        sizeCheckCC<VertexId, Value, SizeT, INSTRUMENT,
                    true > (output, parameter);
    else
        sizeCheckCC<VertexId, Value, SizeT, INSTRUMENT,
                    false> (output, parameter);
}

/**
 * @brief Run test
 *
 * @tparam VertexId Vertex identifier type
 * @tparam Value    Attribute type
 * @tparam SizeT    Graph size type
 *
 * @praam[out] output    Pointer to output graph structure of the problem
 * @param[in]  parameter primitive-specific test parameters
 */
template <
    typename      VertexId,
    typename      Value,
    typename      SizeT >
void instrumentedCC(GRGraph* output, Test_Parameter *parameter)
{
    if (parameter->instrumented)
        debugCC<VertexId, Value, SizeT,  true>(output, parameter);
    else
        debugCC<VertexId, Value, SizeT, false>(output, parameter);
}

/**
 * @brief Run test
 *
 * @tparam VertexId   Vertex identifier type
 * @tparam Value      Attribute type
 * @tparam SizeT      Graph size type
 * @tparam INSTRUMENT Keep kernels statics
 * @tparam DEBUG      Keep debug statics
 * @tparam SIZE_CHECK Enable size check
 *
 * @praam[out] output    Pointer to output graph structure of the problem
 * @param[in]  parameter primitive-specific test parameters
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT,
    bool DEBUG,
    bool SIZE_CHECK >
void runCC(GRGraph* output, Test_Parameter *parameter)
{
    typedef CCProblem < VertexId,
            SizeT,
            Value,
            false > CcProblem; // use double buffer

    typedef CCEnactor < CcProblem,
            INSTRUMENT,
            DEBUG,
            SIZE_CHECK > CcEnactor;

    Csr<VertexId, Value, SizeT> *graph =
        (Csr<VertexId, Value, SizeT>*)parameter->graph;
    bool          quiet              = parameter -> g_quiet;
    int           max_grid_size      = parameter -> max_grid_size;
    int           num_gpus           = parameter -> num_gpus;
    double        max_queue_sizing   = parameter -> max_queue_sizing;
    double        max_in_sizing      = parameter -> max_in_sizing;
    ContextPtr   *context            = (ContextPtr*)parameter -> context;
    std::string   partition_method   = parameter -> partition_method;
    int          *gpu_idx            = parameter -> gpu_idx;
    hipStream_t *streams            = parameter -> streams;
    float         partition_factor   = parameter -> partition_factor;
    int           partition_seed     = parameter -> partition_seed;
    bool          g_stream_from_host = parameter -> g_stream_from_host;
    size_t       *org_size           = new size_t  [num_gpus];
    // Allocate host-side label array
    VertexId    *h_component_ids     = new VertexId[graph->nodes];

    for (int gpu = 0; gpu < num_gpus; gpu++)
    {
        size_t dummy;
        hipSetDevice(gpu_idx[gpu]);
        hipMemGetInfo(&(org_size[gpu]), &dummy);
    }

    CcEnactor* enactor = new CcEnactor(num_gpus, gpu_idx);  // CC enactor map
    CcProblem* problem = new CcProblem;  // Allocate problem on GPU

    util::GRError(
        problem->Init(
            g_stream_from_host,
            graph,
            NULL,
            num_gpus,
            gpu_idx,
            partition_method,
            streams,
            max_queue_sizing,
            max_in_sizing,
            partition_factor,
            partition_seed),
        "CC Problem Initialization Failed", __FILE__, __LINE__);
    util::GRError(
        enactor->Init(context, problem, max_grid_size),
        "CC Enactor Init failed", __FILE__, __LINE__);

    // Perform CC
    CpuTimer cpu_timer;

    util::GRError(
        problem->Reset(enactor->GetFrontierType(), max_queue_sizing),
        "CC Problem Data Reset Failed", __FILE__, __LINE__);
    util::GRError(
        enactor->Reset(), "CC Enactor Reset failed", __FILE__, __LINE__);

    cpu_timer.Start();
    util::GRError(
        enactor->Enact(), "CC Problem Enact Failed", __FILE__, __LINE__);
    cpu_timer.Stop();

    float elapsed = cpu_timer.ElapsedMillis();

    // Copy out results
    util::GRError(
        problem->Extract(h_component_ids),
        "CC Problem Data Extraction Failed", __FILE__, __LINE__);

    unsigned int num_components = problem->num_components;
    output->aggregation = (unsigned int*)&num_components;
    output->node_value1 = (VertexId*)&h_component_ids[0];

    if (!quiet)
    {
        printf(" GPU Connected Component finished in %lf msec.\n", elapsed);
    }

    // Clean up
    if (org_size) { delete[] org_size; org_size = NULL; }
    if (problem ) { delete   problem ; problem  = NULL; }
    if (enactor ) { delete   enactor ; enactor  = NULL; }
}

/**
 * @brief Dispatch function to handle configurations
 *
 * @param[out] grapho  Pointer to output graph structure of the problem
 * @param[in]  graphi  Pointer to input graph we need to process on
 * @param[in]  config  Primitive-specific configurations
 * @param[in]  data_t  Data type configurations
 * @param[in]  context ModernGPU context
 * @param[in]  streams CUDA stream
 */
void dispatch_cc(
    GRGraph*       grapho,
    const GRGraph* graphi,
    const GRSetup  config,
    const GRTypes  data_t,
    ContextPtr*    context,
    hipStream_t*  streams)
{
    Test_Parameter *parameter = new Test_Parameter;
    parameter->context  = context;
    parameter->streams  = streams;
    parameter->g_quiet  = config.quiet;
    parameter->num_gpus = config.num_devices;
    parameter->gpu_idx  = config.device_list;

    switch (data_t.VTXID_TYPE)
    {
    case VTXID_INT:
    {
        switch (data_t.SIZET_TYPE)
        {
        case SIZET_INT:
        {
            switch (data_t.VALUE_TYPE)
            {
            case VALUE_INT:    // template type = <int, int, int>
            {
                // build input CSR format graph
                Csr<int, int, int> csr(false);
                csr.nodes = graphi->num_nodes;
                csr.edges = graphi->num_edges;
                csr.row_offsets    = (int*)graphi->row_offsets;
                csr.column_indices = (int*)graphi->col_indices;
                parameter->graph = &csr;

                instrumentedCC<int, int, int>(grapho, parameter);

                // reset for free memory
                csr.row_offsets    = NULL;
                csr.column_indices = NULL;
                break;
            }
            case VALUE_UINT:    // template type = <int, uint, int>
            {
                printf("Not Yet Support This DataType Combination.\n");
                break;
            }
            case VALUE_FLOAT:    // template type = <int, float, int>
            {
                printf("Not Yet Support This DataType Combination.\n");
                break;
            }
            }
            break;
        }
        }
        break;
    }
    }
}

/*
 * @brief Entry of gunrock_cc function
 *
 * @param[out] grapho Pointer to output graph structure of the problem
 * @param[in]  graphi Pointer to input graph we need to process on
 * @param[in]  config Gunrock primitive specific configurations
 * @param[in]  data_t Gunrock data type structure
 */
void gunrock_cc(
    GRGraph       *grapho,
    const GRGraph *graphi,
    const GRSetup  config,
    const GRTypes  data_t)
{
    // GPU-related configurations
    int           num_gpus =    0;
    int           *gpu_idx = NULL;
    ContextPtr    *context = NULL;
    hipStream_t  *streams = NULL;

    num_gpus = config.num_devices;
    gpu_idx  = new int [num_gpus];
    for (int i = 0; i < num_gpus; ++i)
    {
        gpu_idx[i] = config.device_list[i];
    }

    // Create streams and MordernGPU context for each GPU
    streams = new hipStream_t[num_gpus * num_gpus * 2];
    context = new ContextPtr[num_gpus * num_gpus];
    if (!config.quiet) { printf(" using %d GPUs:", num_gpus); }
    for (int gpu = 0; gpu < num_gpus; ++gpu)
    {
        if (!config.quiet) { printf(" %d ", gpu_idx[gpu]); }
        util::SetDevice(gpu_idx[gpu]);
        for (int i = 0; i < num_gpus * 2; ++i)
        {
            int _i = gpu * num_gpus * 2 + i;
            util::GRError(hipStreamCreate(&streams[_i]),
                          "hipStreamCreate fialed.", __FILE__, __LINE__);
            if (i < num_gpus)
            {
                context[gpu * num_gpus + i] =
                    mgpu::CreateCudaDeviceAttachStream(gpu_idx[gpu],
                                                       streams[_i]);
            }
        }
    }
    if (!config.quiet) { printf("\n"); }

    dispatch_cc(grapho, graphi, config, data_t, context, streams);
}

/*
 * @brief Simple interface take in CSR arrays as input
 *
 * @param[out] components  Return component ID for each node
 * @param[out] num_comps   Return number of components calculated
 * @param[in]  num_nodes   Number of nodes of the input graph
 * @param[in]  num_edges   Number of edges of the input graph
 * @param[in]  row_offsets CSR-formatted graph input row offsets
 * @param[in]  col_indices CSR-formatted graph input column indices
 */
int cc(
    int*       component,
    const int  num_nodes,
    const int  num_edges,
    const int* row_offsets,
    const int* col_indices)
{
    struct GRTypes data_t;          // primitive-specific data types
    data_t.VTXID_TYPE = VTXID_INT;  // integer vertex identifier
    data_t.SIZET_TYPE = SIZET_INT;  // integer graph size type
    data_t.VALUE_TYPE = VALUE_INT;  // integer attributes type

    struct GRSetup config = InitSetup();  // primitive-specific configures

    struct GRGraph *grapho = (struct GRGraph*)malloc(sizeof(struct GRGraph));
    struct GRGraph *graphi = (struct GRGraph*)malloc(sizeof(struct GRGraph));

    graphi->num_nodes   = num_nodes;  // setting graph nodes
    graphi->num_edges   = num_edges;  // setting graph edges
    graphi->row_offsets = (void*)&row_offsets[0];  // setting row_offsets
    graphi->col_indices = (void*)&col_indices[0];  // setting col_indices
    printf(" loaded %d nodes and %d edges\n", num_nodes, num_edges);

    gunrock_cc(grapho, graphi, config, data_t);
    int* num_components = (int*)grapho->aggregation;
    memcpy(component, (int*)grapho->node_value1, num_nodes * sizeof(int));

    if (graphi) free(graphi);
    if (grapho) free(grapho);

    return *num_components;
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:

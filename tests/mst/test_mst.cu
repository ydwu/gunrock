// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file
 * test_mst.cu
 *
 * @brief Simple test driver for computing Minimum Spanning Tree.
 */

#include <stdio.h>
#include <vector>
#include <utility>
#include <iostream>
#include <cstdlib>
#include <algorithm>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph construction utilities
#include <gunrock/graphio/market.cuh>

// MST includes
#include <gunrock/app/cc/cc_app.cu>
#include <gunrock/app/mst/mst_enactor.cuh>
#include <gunrock/app/mst/mst_problem.cuh>
#include <gunrock/app/mst/mst_functor.cuh>

// Operator includes
#include <gunrock/oprtr/advance/kernel.cuh>
#include <gunrock/oprtr/filter/kernel.cuh>
#include <moderngpu.cuh>

// CPU Kruskal MST reference
#include <boost/config.hpp>
#include <boost/graph/adjacency_list.hpp>
#include <boost/graph/kruskal_min_spanning_tree.hpp>

using namespace gunrock;
using namespace gunrock::app;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::mst;

///////////////////////////////////////////////////////////////////////////////
// Housekeeping and utility routines
///////////////////////////////////////////////////////////////////////////////

void Usage()
{
    printf(
        " ------------------------------------------------------------------\n"
        " test_mst <graph type> <graph type args> [--device=<device_index>]\n"
        " [--quick] [--quiet] [--v]\n\n"
        "Graph types and args:\n"
        "  market [<file>]\n"
        "    Reads a Matrix-Market coordinate-format graph of directed/undirected\n"
        "    edges from STDIN (or from the optionally-specified file)\n"
        "  --device=<device_index> Set GPU device for running the graph primitive\n"
        "  --quick If set will skip the CPU validation code\n"
        "  --quiet If set don't print anything\n"
        "  --v If set will enable debug mode\n\n"
        " ------------------------------------------------------------------\n");
}

/**
 * @brief Displays the MST result.
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph.
 * @param[in] edge_mask Pointer to the MST edge mask.
 */
template<typename VertexId, typename Value, typename SizeT>
void DisplaySolution(
    const Csr<VertexId, Value, SizeT> &graph, int *edge_mask)
{
    int count = 0;
    int print_limit = graph.nodes;
    if (print_limit > 10)
    {
        print_limit = 10;
    }

    // find source vertex ids for display results
    VertexId *source = new VertexId[graph.edges];
    for (int i = 0; i < graph.nodes; ++i)
    {
        for (int j = graph.row_offsets[i]; j < graph.row_offsets[i + 1]; ++j)
        {
            source[j] = i;
        }
    }

    // print source-destination pairs of minimum spanning tree edges
    printf("GPU Minimum Spanning Tree [First %d edges]\n", print_limit);
    printf("src dst\n");
    for (int i = 0; i < graph.edges; ++i)
    {
        if (edge_mask[i] == 1 && count <= print_limit)
        {
            printf("%d %d\n", source[i], graph.column_indices[i]);
            ++count;
        }
    }

    // clean up if necessary
    if (source) { delete [] source; }
}

///////////////////////////////////////////////////////////////////////////////
// CPU validation routines
///////////////////////////////////////////////////////////////////////////////

/**
 * @brief A simple CPU-based reference MST implementation.
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] edge_values Weight value associated with each edge.
 * @param[in] graph Reference to the CSR graph we process on.
 *
 *  \return long long int which indicates the total weight of the graph.
 */
template<typename VertexId, typename Value, typename SizeT>
Value SimpleReferenceMST(
    const Value *edge_values,
    const Csr<VertexId, Value, SizeT> &graph,
    bool quiet_mode = false)
{
    if (!quiet_mode) { printf("\nMST CPU REFERENCE TEST\n"); }

    // Kruskal's minimum spanning tree preparations
    using namespace boost;
    typedef adjacency_list< vecS, vecS, undirectedS,
            no_property, property<edge_weight_t, int> > Graph;
    typedef graph_traits < Graph >::edge_descriptor   Edge;
    typedef graph_traits < Graph >::vertex_descriptor Vertex;
    typedef std::pair<VertexId, VertexId> E;

    E *edge_pairs = new E[graph.edges];
    int idx = 0;
    for (int i = 0; i < graph.nodes; ++i)
    {
        for (int j = graph.row_offsets[i]; j < graph.row_offsets[i + 1]; ++j)
        {
            edge_pairs[idx++] = std::make_pair(i, graph.column_indices[j]);
        }
    }

    Graph g(edge_pairs, edge_pairs + graph.edges, edge_values, graph.nodes);
    property_map < Graph, edge_weight_t >::type weight = get(edge_weight, g);
    std::vector < Edge > spanning_tree;

    CpuTimer cpu_timer; // record the kernel running time
    cpu_timer.Start();

    // compute reference using kruskal_min_spanning_tree algorithm
    kruskal_minimum_spanning_tree(g, std::back_inserter(spanning_tree));

    cpu_timer.Stop();
    float elapsed_cpu = cpu_timer.ElapsedMillis();

    // analyze reference results
    SizeT num_selected_cpu = 0;
    Value total_weight_cpu = 0;

    if (graph.nodes <= 50 && !quiet_mode)
    {
        printf("CPU Minimum Spanning Tree\n");
    }
    for (std::vector < Edge >::iterator ei = spanning_tree.begin();
            ei != spanning_tree.end(); ++ei)
    {
        if (graph.nodes <= 50 && !quiet_mode)
        {
            // print the edge pairs in the minimum spanning tree
            printf("%ld %ld\n", source(*ei, g), target(*ei, g));
            // printf("  with weight of %f\n", weight[*ei]);
        }
        ++num_selected_cpu;
        total_weight_cpu += weight[*ei];
    }

    // clean up if necessary
    if (edge_pairs) { delete [] edge_pairs; }

    if (!quiet_mode)
    {
        printf("CPU - Computation Complete in %lf msec.\n", elapsed_cpu);
        // printf("CPU - Number of Edges in MST: %d\n", num_selected_cpu);
    }

    return total_weight_cpu;
}

///////////////////////////////////////////////////////////////////////////////
// GPU MST test routines
///////////////////////////////////////////////////////////////////////////////

/**
 * @brief Test entry
 *
 * @tparam VertexId
 * @tparam SizeT
 * @tparam Value
 * @tparam DEBUG
 * @tparam SIZE_CHECK
 */
template <
    typename VertexId,
    typename SizeT,
    typename Value,
    bool     DEBUG,
    bool     SIZE_CHECK >
void RunTest(Info<VertexId, Value, SizeT> *info)
{
    // define the problem data structure for graph primitive
    typedef MSTProblem<VertexId,
            SizeT,
            Value,
            true,    // MARK_PREDECESSORS
            false,   // ENABLE_IDEMPOTENCE
            true >   // USE_DOUBLE_BUFFER
            Problem;

    Csr<VertexId, Value, SizeT>* graph =
        (Csr<VertexId, Value, SizeT>*)info->csr_ptr;
    int num_gpus            = info->info["num_gpus"].get_int();
    int max_grid_size       = info->info["max_grid_size"].get_int();
    int iterations          = info->info["num_iteration"].get_int();
    bool quiet_mode         = info->info["quiet_mode"].get_bool();
    bool quick_mode         = info->info["quick_mode"].get_bool();
    bool stream_from_host   = info->info["stream_from_host"].get_bool();
    double max_queue_sizing = info->info["max_queue_sizing"].get_real();
    json_spirit::mArray device_list = info->info["device_list"].get_array();
    int* gpu_idx = new int[num_gpus];
    for (int i = 0; i < num_gpus; i++) gpu_idx[i] = device_list[i].get_int();

    // TODO: remove after merge mgpu-cq
    ContextPtr* context = (ContextPtr*)info->context;

    // allocate MST enactor map
    MSTEnactor < Problem,
               false,        // INSTRUMENT
               DEBUG,        // DEBUG
               SIZE_CHECK >  // SIZE_CHECK
               enactor(gpu_idx);

    // allocate problem on GPU create a pointer of the MSTProblem type
    Problem * problem = new Problem;

    // host results spaces
    VertexId * edge_mask = new VertexId[graph->edges];

    if (!quiet_mode) { printf("\nMINIMUM SPANNING TREE TEST\n"); }

    // copy data from CPU to GPU initialize data members in DataSlice
    util::GRError(problem->Init(stream_from_host, *graph, num_gpus),
                  "Problem MST Initialization Failed", __FILE__, __LINE__);

    // perform calculations
    GpuTimer gpu_timer;  // record the kernel running time
    double elapsed_gpu = 0.0f;  // device elapsed running time

    for (int iter = 0; iter < iterations; ++iter)
    {
        // reset values in DataSlice
        util::GRError(problem->Reset(
                          enactor.GetFrontierType(), max_queue_sizing),
                      "MST Problem Data Reset Failed", __FILE__, __LINE__);

        gpu_timer.Start();

        // launch MST enactor
        util::GRError(enactor.template Enact<Problem>(
                          *context, problem, max_grid_size),
                      "MST Problem Enact Failed", __FILE__, __LINE__);

        gpu_timer.Stop();
        elapsed_gpu += gpu_timer.ElapsedMillis();
    }

    elapsed_gpu /= iterations;
    if (!quiet_mode)
    {
        printf("GPU - Computation Complete in %lf msec.\n", elapsed_gpu);
    }

    // copy results back to CPU from GPU using Extract
    util::GRError(problem->Extract(edge_mask),
                  "MST Problem Data Extraction Failed", __FILE__, __LINE__);

    if (!quick_mode)  // run CPU reference test
    {
        // calculate GPU final number of selected edges
        int num_selected_gpu = 0;
        for (int iter = 0; iter < graph->edges; ++iter)
        {
            num_selected_gpu += edge_mask[iter];
        }
        // printf("\nGPU - Number of Edges in MST: %d\n", num_selected_gpu);

        // calculate GPU total selected MST weights for validation
        Value total_weight_gpu = 0;
        for (int iter = 0; iter < graph->edges; ++iter)
        {
            total_weight_gpu += edge_mask[iter] * graph->edge_values[iter];
        }

        // correctness validation
        Value total_weight_cpu = SimpleReferenceMST(
                                     graph->edge_values, *graph, quiet_mode);
        if (total_weight_cpu == total_weight_gpu)
        {
            // print the edge pairs in the minimum spanning tree
            if (!quiet_mode) DisplaySolution(*graph, edge_mask);
            if (!quiet_mode) { printf("\nCORRECT.\n"); }
        }
        else
        {
            if (!quiet_mode)
            {
                printf("INCORRECT.\n");
                std::cout << "CPU Weight = " << total_weight_cpu << std::endl;
                std::cout << "GPU Weight = " << total_weight_gpu << std::endl;
            }
        }
    }


    info->ComputeCommonStats(enactor.enactor_stats.GetPointer(), elapsed_gpu);

    if (!quiet_mode)
    {
        info->DisplayStats(false);   // display collected statistics
    }

    info->CollectInfo();

    // clean up if necessary
    if (problem)   delete    problem;
    if (edge_mask) delete [] edge_mask;
}

/**
 * @brief Test entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam DEBUG
 *
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool     DEBUG >
void RunTests_size_check(Info<VertexId, Value, SizeT> *info)
{
    if (info->info["size_check"].get_bool())
    {
        RunTest <VertexId, Value, SizeT, DEBUG,  true>(info);
    }
    else
    {
        RunTest <VertexId, Value, SizeT, DEBUG, false>(info);
    }
}

/**
 * @brief Test entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT >
void RunTests_debug(Info<VertexId, Value, SizeT> *info)
{
    if (info->info["debug_mode"].get_bool())
    {
        RunTests_size_check <VertexId, Value, SizeT,  true>(info);
    }
    else
    {
        RunTests_size_check <VertexId, Value, SizeT, false>(info);
    }
}

/**
 * @brief Test entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 */
template <typename VertexId, typename Value, typename SizeT>
void RunTest_connectivity_check(Info<VertexId, Value, SizeT> *info)
{
    // test graph connectivity because MST only supports fully-connected graph
    struct GRTypes data_t;          // data type structure
    data_t.VTXID_TYPE = VTXID_INT;  // vertex identifier
    data_t.SIZET_TYPE = SIZET_INT;  // graph size type
    data_t.VALUE_TYPE = VALUE_INT;  // attributes type

    struct GRSetup config = InitSetup();  // gunrock configurations

    struct GRGraph *grapho = (GRGraph*)malloc(sizeof(GRGraph));
    struct GRGraph *graphi = (GRGraph*)malloc(sizeof(GRGraph));

    graphi->num_nodes = info->csr_ptr->nodes;
    graphi->num_edges = info->csr_ptr->edges;
    graphi->row_offsets = (void*)&info->csr_ptr->row_offsets[0];
    graphi->col_indices = (void*)&info->csr_ptr->column_indices[0];

    gunrock_cc(grapho, graphi, config, data_t);

    // run test only if the graph is fully-connected
    int* num_cc = (int*)grapho->aggregation;
    if (*num_cc == 1)  // perform minimum spanning tree test
    {
        RunTests_debug<VertexId, Value, SizeT>(info);
    }
    else  // more than one connected components in the graph
    {
        fprintf(stderr, "Unsupported non-fully connected graph input.\n");
        exit(1);
    }

    if (graphi) free(graphi);
    if (grapho) free(grapho);
}

///////////////////////////////////////////////////////////////////////////////
// Main function
///////////////////////////////////////////////////////////////////////////////

int main(int argc, char** argv)
{
    CommandLineArgs args(argc, argv);
    int graph_args = argc - args.ParsedArgc() - 1;
    if (argc < 2 || graph_args < 1 || args.CheckCmdLineFlag("help"))
    {
        Usage();
        return 1;
    }

    typedef int VertexId;  // use int as the vertex identifier
    typedef int Value;     // use int as the value type
    typedef int SizeT;     // use int as the graph size type

    Csr<VertexId, Value, SizeT> csr(false);  // graph we process on
    Info<VertexId, Value, SizeT> *info = new Info<VertexId, Value, SizeT>;

    // graph construction or generation related parameters
    info->info["undirected"] = true;  // always convert to undirected
    info->info["edge_value"] = true;  // require per edge weight values

    info->Init("MST", args, csr);
    RunTest_connectivity_check<VertexId, Value, SizeT>(info);  // run test

    return 0;
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End

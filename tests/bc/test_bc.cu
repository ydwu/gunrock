// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_bc.cu
 *
 * @brief Simple test driver program for BC.
 */

#include <stdio.h>
#include <string>
#include <deque>
#include <vector>
#include <queue>
#include <iostream>
#include <fstream>
#include <algorithm>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// BC includes
#include <gunrock/app/bc/bc_enactor.cuh>
#include <gunrock/app/bc/bc_problem.cuh>
#include <gunrock/app/bc/bc_functor.cuh>

// Operator includes
#include <gunrock/oprtr/advance/kernel.cuh>
#include <gunrock/oprtr/filter/kernel.cuh>

#include <moderngpu.cuh>

// Boost includes
#include <boost/config.hpp>
#include <boost/graph/adjacency_list.hpp>
#include <boost/graph/connected_components.hpp>
#include <boost/graph/bc_clustering.hpp>
#include <boost/graph/iteration_macros.hpp>

using namespace gunrock;
using namespace gunrock::app;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::bc;

/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/
void Usage()
{
    printf(
        "\ntest_bc <graph type> <graph type args> [--device=<device_index>] "
        "[--instrumented] [--src=<source index>] [--quick] [--v]"
        "[--queue-sizing=<scale factor>] [--ref-file=<reference filename>]\n"
        "[--in-sizing=<in/out queue scale factor>] [--disable-size-check] "
        "[--grid-size=<grid size>] [partition_method=random / biasrandom / clustered / metis]\n"
        "\n"
        "Graph types and args:\n"
        "  market [<file>]\n"
        "    Reads a Matrix-Market coordinate-formatted graph of undirected\n"
        "    edges from stdin (or from the optionally-specified file).\n"
        "--device=<device_index>: Set GPU device for running the graph primitive.\n"
        "--undirected: If set then treat the graph as undirected graph.\n"
        "--instrumented: If set then kernels keep track of queue-search_depth\n"
        "and barrier duty (a relative indicator of load imbalance.)\n"
        "--src=<source index>: When source index is -1, compute BC value for each\n"
        "node. Otherwise, debug the delta value for one node\n"
        "--quick: If set will skip the CPU validation code.\n"
        "--queue-sizing Allocates a frontier queue sized at (graph-edges * <scale factor>).\n"
        "Default is 1.0.\n"
        "--v: If set, enable verbose output, keep track of the kernel running.\n"
        "--ref-file: If set, use pre-computed result stored in ref-file to verify.\n"
    );
}

/**
 * @brief Displays the BC result (sigma value and BC value)
 *
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] sigmas
 * @param[in] bc_values
 * @param[in] nodes
 */
template<typename Value, typename SizeT>
void DisplaySolution(Value *sigmas, Value *bc_values, SizeT nodes)
{
    if (nodes < 40)
    {
        printf("[");
        for (SizeT i = 0; i < nodes; ++i)
        {
            PrintValue(i);
            printf(":");
            PrintValue(sigmas[i]);
            printf(",");
            PrintValue(bc_values[i]);
            printf(" ");
        }
        printf("]\n");
    }
}

/******************************************************************************
* BC Testing Routines
*****************************************************************************/

/**
 * @brief Graph edge properties (bundled properties)
 */
struct EdgeProperties
{
    int weight;
};

/**
 * @brief A simple CPU-based reference BC ranking implementation.
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to graph we process on
 * @param[in] bc_values Pointer to node bc value
 * @param[in] ebc_values Pointer to edge bc value
 * @param[in] sigmas Pointer to node sigma value
 * @param[in] src VertexId of source node if there is any
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT >
void ReferenceBC(
    const Csr<VertexId, Value, SizeT> &graph,
    Value                             *bc_values,
    Value                             *ebc_values,
    Value                             *sigmas,
    VertexId                          *source_path,
    VertexId                           src,
    bool                               quiet = false)
{
    typedef Coo<VertexId, Value> EdgeTupleType;
    EdgeTupleType *coo = (EdgeTupleType*) malloc(sizeof(EdgeTupleType) * graph.edges);
    if (src == -1)
    {
        // Perform full exact BC using BGL

        using namespace boost;
        typedef adjacency_list <setS, vecS, undirectedS, no_property,
                EdgeProperties> Graph;
        typedef Graph::vertex_descriptor Vertex;
        typedef Graph::edge_descriptor Edge;

        Graph G;
        for (int i = 0; i < graph.nodes; ++i)
        {
            for (int j = graph.row_offsets[i]; j < graph.row_offsets[i + 1]; ++j)
            {
                add_edge(vertex(i, G), vertex(graph.column_indices[j], G), G);
            }
        }

        typedef std::map<Edge, int> StdEdgeIndexMap;
        StdEdgeIndexMap my_e_index;
        typedef boost::associative_property_map< StdEdgeIndexMap > EdgeIndexMap;
        EdgeIndexMap e_index(my_e_index);

        int i = 0;
        BGL_FORALL_EDGES(edge, G, Graph)
        {
            my_e_index.insert(std::pair<Edge, int>(edge, i));
            ++i;
        }

        // Define EdgeCentralityMap
        std::vector< double > e_centrality_vec(boost::num_edges(G), 0.0);
        // Create the external property map
        boost::iterator_property_map< std::vector< double >::iterator,
              EdgeIndexMap >
              e_centrality_map(e_centrality_vec.begin(), e_index);

        // Define VertexCentralityMap
        typedef boost::property_map< Graph, boost::vertex_index_t>::type
        VertexIndexMap;
        VertexIndexMap v_index = get(boost::vertex_index, G);
        std::vector< double > v_centrality_vec(boost::num_vertices(G), 0.0);

        // Create the external property map
        boost::iterator_property_map< std::vector< double >::iterator,
              VertexIndexMap>
              v_centrality_map(v_centrality_vec.begin(), v_index);

        // Perform BC
        CpuTimer cpu_timer;
        cpu_timer.Start();
        brandes_betweenness_centrality(G, v_centrality_map, e_centrality_map);
        cpu_timer.Stop();
        float elapsed = cpu_timer.ElapsedMillis();

        BGL_FORALL_VERTICES(vertex, G, Graph)
        {
            bc_values[vertex] = (Value)v_centrality_map[vertex];
        }

        int idx = 0;
        BGL_FORALL_EDGES(edge, G, Graph)
        {
            coo[idx].row = source(edge, G);
            coo[idx].col = target(edge, G);
            coo[idx++].val = (Value)e_centrality_map[edge];
            coo[idx].col = source(edge, G);
            coo[idx].row = target(edge, G);
            coo[idx++].val = (Value)e_centrality_map[edge];
        }

        std::stable_sort(coo, coo + graph.edges,
                         RowFirstTupleCompare<EdgeTupleType>);

        for (idx = 0; idx < graph.edges; ++idx)
        {
            //std::cout << coo[idx].row << "," << coo[idx].col
            //          << ":" << coo[idx].val << std::endl;
            //ebc_values[idx] = coo[idx].val;
        }

        if (!quiet)
        {
            printf("CPU BC finished in %lf msec.", elapsed);
        }
    }
    else
    {
        //Simple BFS pass to get single pass BC
        //VertexId *source_path = new VertexId[graph.nodes];

        //initialize distances
        for (VertexId i = 0; i < graph.nodes; ++i)
        {
            source_path[i] = -1;
            bc_values[i] = 0;
            sigmas[i] = 0;
        }
        source_path[src] = 0;
        VertexId search_depth = 0;
        sigmas[src] = 1;

        // Initialize queue for managing previously-discovered nodes
        std::deque<VertexId> frontier;
        frontier.push_back(src);

        //
        //Perform one pass of BFS for one source
        //

        CpuTimer cpu_timer;
        cpu_timer.Start();
        while (!frontier.empty())
        {

            // Dequeue node from frontier
            VertexId dequeued_node = frontier.front();
            frontier.pop_front();
            VertexId neighbor_dist = source_path[dequeued_node] + 1;

            // Locate adjacency list
            int edges_begin = graph.row_offsets[dequeued_node];
            int edges_end = graph.row_offsets[dequeued_node + 1];

            for (int edge = edges_begin; edge < edges_end; ++edge)
            {
                // Lookup neighbor and enqueue if undiscovered
                VertexId neighbor = graph.column_indices[edge];
                if (source_path[neighbor] == -1)
                {
                    source_path[neighbor] = neighbor_dist;
                    sigmas[neighbor] += sigmas[dequeued_node];
                    if (search_depth < neighbor_dist)
                    {
                        search_depth = neighbor_dist;
                    }

                    frontier.push_back(neighbor);
                }
                else
                {
                    if (source_path[neighbor] == source_path[dequeued_node] + 1)
                        sigmas[neighbor] += sigmas[dequeued_node];
                }
            }
        }
        search_depth++;

        for (int iter = search_depth - 2; iter > 0; --iter)
        {

            int cur_level = 0;
            for (int node = 0; node < graph.nodes; ++node)
            {
                if (source_path[node] == iter)
                {
                    ++cur_level;
                    int edges_begin = graph.row_offsets[node];
                    int edges_end = graph.row_offsets[node + 1];

                    for (int edge = edges_begin; edge < edges_end; ++edge)
                    {
                        VertexId neighbor = graph.column_indices[edge];
                        if (source_path[neighbor] == iter + 1)
                        {
                            bc_values[node] +=
                                1.0f * sigmas[node] / sigmas[neighbor] *
                                (1.0f + bc_values[neighbor]);
                        }
                    }
                }
            }
        }

        for (int i = 0; i < graph.nodes; ++i)
        {
            bc_values[i] *= 0.5f;
        }

        cpu_timer.Stop();
        float elapsed = cpu_timer.ElapsedMillis();

        if (!quiet)
        {
            printf("CPU BC finished in %lf msec. Search depth: %d\n",
                   elapsed, search_depth);
        }

        //delete[] source_path;
    }
    free(coo);
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 * @tparam DEBUG
 * @tparam SIZE_CHECK
 *
 * @param[in] info Pointer to mObject info.
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT,
    bool DEBUG,
    bool SIZE_CHECK >
void RunTests(Info<VertexId, Value, SizeT> *info)
{
    typedef BCProblem < VertexId,
            SizeT,
            Value,
            true,   // MARK_PREDECESSORS
            false > BcProblem;  //does not use double buffer

    typedef BCEnactor < BcProblem,
            INSTRUMENT,
            DEBUG,
            SIZE_CHECK >
            BcEnactor;

    // parse configurations from mObject info
    Csr<VertexId, Value, SizeT> *graph = info->csr_ptr;
    VertexId src                 = info->info["source_vertex"].get_int64();
    bool quiet_mode              = info->info["quiet_mode"].get_bool();
    int max_grid_size            = info->info["max_grid_size"].get_int();
    int num_gpus                 = info->info["num_gpus"].get_int();
    double max_queue_sizing      = info->info["max_queue_sizing"].get_real();
    double max_queue_sizing1     = info->info["max_queue_sizing1"].get_real();
    double max_in_sizing         = info->info["max_in_sizing"].get_real();
    std::string partition_method = info->info["partition_method"].get_str();
    double partition_factor      = info->info["partition_factor"].get_real();
    int partition_seed           = info->info["partition_seed"].get_int();
    bool quick_mode              = info->info["quick_mode"].get_bool();
    bool stream_from_host        = info->info["stream_from_host"].get_bool();
    int iterations               = info->info["num_iteration"].get_int();
    std::string ref_filename     = info->info["ref_filename"].get_str();

    json_spirit::mArray device_list = info->info["device_list"].get_array();
    int* gpu_idx = new int[num_gpus];
    for (int i = 0; i < num_gpus; i++) gpu_idx[i] = device_list[i].get_int();

    // TODO: remove after merge mgpu-cq
    ContextPtr   *context = (ContextPtr*)  info->context;
    hipStream_t *streams = (hipStream_t*)info->streams;

    // Allocate host-side array (for both reference and GPU-computed results)
    Value        *reference_bc_values        = new Value   [graph->nodes];
    Value        *reference_ebc_values       = new Value   [graph->edges];
    Value        *reference_sigmas           = new Value   [graph->nodes];
    VertexId     *reference_labels           = new VertexId[graph->nodes];
    Value        *h_sigmas                   = new Value   [graph->nodes];
    Value        *h_bc_values                = new Value   [graph->nodes];
    Value        *h_ebc_values               = new Value   [graph->edges];
    VertexId     *h_labels                   = new VertexId[graph->nodes];
    Value        *reference_check_bc_values  = (quick_mode)                ? NULL : reference_bc_values;
    Value        *reference_check_ebc_values = (quick_mode || (src != -1)) ? NULL : reference_ebc_values;
    Value        *reference_check_sigmas     = (quick_mode || (src == -1)) ? NULL : reference_sigmas;
    VertexId     *reference_check_labels     = (quick_mode || (src == -1)) ? NULL : reference_labels;

    size_t *org_size = new size_t[num_gpus];
    for (int gpu = 0; gpu < num_gpus; gpu++)
    {
        size_t dummy;
        hipSetDevice(gpu_idx[gpu]);
        hipMemGetInfo(&(org_size[gpu]), &dummy);
    }

    BcEnactor* enactor = new BcEnactor(num_gpus, gpu_idx);  // enactor map
    BcProblem* problem = new BcProblem;  // allocate problem on GPU

    util::GRError(problem->Init(
                      stream_from_host,
                      graph,
                      NULL,
                      num_gpus,
                      gpu_idx,
                      partition_method,
                      streams,
                      max_queue_sizing,
                      max_in_sizing,
                      partition_factor,
                      partition_seed),
                  "BC Problem Initialization Failed", __FILE__, __LINE__);

    util::GRError(enactor->Init(context, problem, max_grid_size),
                  "BC Enactor init failed", __FILE__, __LINE__);

    // compute reference CPU BC solution for source-distance
    if (reference_check_bc_values != NULL)
    {
        if (ref_filename.empty())
        {
            if (!quiet_mode) { printf("Computing reference value ...\n"); }
            ReferenceBC(
                *graph,
                reference_check_bc_values,
                reference_check_ebc_values,
                reference_check_sigmas,
                reference_check_labels,
                src,
                quiet_mode);
            if (!quiet_mode) { printf("\n"); }
        }
        else
        {
            std::ifstream fin;
            fin.open(ref_filename.c_str(), std::ios::binary);
            for (int i = 0; i < graph->nodes; ++i)
            {
                fin.read(reinterpret_cast<char*>(&reference_check_bc_values[i]), sizeof(Value));
            }
            fin.close();
        }
    }

    // perform BC
    double elapsed  = 0.0f;
    CpuTimer cpu_timer;

    VertexId start_src, end_src;
    if (src == -1)
    {
        start_src = 0;
        end_src = graph->nodes;
    }
    else
    {
        start_src = src;
        end_src = src + 1;
    }

    for (int iter = 0; iter < iterations; ++iter)
    {
        for (int gpu = 0; gpu < num_gpus; gpu++)
        {
            util::SetDevice(gpu_idx[gpu]);
            util::MemsetKernel <<< 128, 128>>>(
                problem->data_slices[gpu]->bc_values.GetPointer(util::DEVICE),
                (Value)0.0f, (int)(problem->sub_graphs[gpu].nodes));
        }
        util::GRError(problem->Reset(
                          0, enactor->GetFrontierType(),
                          max_queue_sizing, max_queue_sizing1),
                      "BC Problem Data Reset Failed", __FILE__, __LINE__);

        if (!quiet_mode)
        {
            printf("__________________________\n"); fflush(stdout);
        }
        cpu_timer.Start();
        for (VertexId i = start_src; i < end_src; ++i)
        {
            util::GRError(problem->Reset(
                              i, enactor->GetFrontierType(),
                              max_queue_sizing, max_queue_sizing1),
                          "BC Problem Data Reset Failed", __FILE__, __LINE__);
            util::GRError(enactor ->Reset(),
                          "BC Enactor Reset failed", __FILE__, __LINE__);
            util::GRError(enactor ->Enact(i),
                          "BC Problem Enact Failed", __FILE__, __LINE__);
        }
        for (int gpu = 0; gpu < num_gpus; gpu++)
        {
            util::SetDevice(gpu_idx[gpu]);
            util::MemsetScaleKernel <<< 128, 128>>>(
                problem->data_slices[gpu]->bc_values.GetPointer(util::DEVICE),
                (Value)0.5f, (int)(problem->sub_graphs[gpu].nodes));
        }
        cpu_timer.Stop();
        if (!quiet_mode)
        {
            printf("--------------------------\n"); fflush(stdout);
        }
        elapsed += cpu_timer.ElapsedMillis();
    }

    elapsed /= iterations;

    // Copy out results
    util::GRError(problem->Extract(
                      h_sigmas, h_bc_values, h_ebc_values, h_labels),
                  "BC Problem Data Extraction Failed", __FILE__, __LINE__);
    /*
    printf("edge bc values: %d\n", graph.edges);
    for (int i = 0; i < graph.edges; ++i) {
        printf("%5f, %5f\n", h_ebc_values[i], reference_check_ebc_values[i]);
    }
    printf("edge bc values end\n");*/

    /*std::queue<VertexId> temp_queue;
    int *temp_marker=new int[graph->nodes];
    memset(temp_marker, 0, sizeof(int)*graph->nodes);
    temp_queue.push(41107);
    temp_queue.push(41109);
    cout<<"parent\tchild\tlabel\tsigma\tbc_value\t\tlabel\tsigma\tbc_value"<<endl;
    while (!temp_queue.empty())
    {
        VertexId parent = temp_queue.front();
        temp_queue.pop();
        temp_marker[parent]=1;
        int      gpu     = problem->partition_tables[0][parent];
        VertexId parent_ = problem->convertion_tables[0][parent];
        util::SetDevice(gpu_idx[gpu]);
        for (int i=graph->row_offsets[parent];i<graph->row_offsets[parent+1];i++)
        {
            VertexId child = graph->column_indices[i];
            VertexId child_ = 0;

            for (int j=problem->graph_slices[gpu]->row_offsets[parent_];
                     j<problem->graph_slices[gpu]->row_offsets[parent_+1];j++)
            {
                VertexId c=problem->graph_slices[gpu]->column_indices[j];
                if (problem->graph_slices[gpu]->original_vertex[c] == child)
                {
                    child_=c;break;
                }
            }
            //if (h_labels[child] != h_labels[parent]+1) continue;
            cout<<parent<<"\t "<<child<<"\t "<<h_labels[child]<<"\t "<<h_sigmas[child]<<"\t "<<h_bc_values[child]<<"\t";
            if (reference_check_labels[child] != h_labels[child] ||
                reference_check_sigmas[child] != h_sigmas[child] ||
                reference_check_bc_values[child] != h_bc_values[child])
            {
                cout<<"*";
                if (h_labels[child]==h_labels[parent]+1 && temp_marker[child]!=1) temp_queue.push(child);
            }
            cout<<"\t "<<reference_check_labels[child]<<"\t "<<reference_check_sigmas[child]<<"\t "<<reference_check_bc_values[child];
            cout<<"\t "<<gpu<<"\t "<<parent_<<"\t "<<child_;
            VertexId temp_label;
            Value    temp_sigma, temp_bc;
            hipMemcpy((void*)&temp_label, problem->data_slices[gpu]->labels.GetPointer(util::DEVICE)+child_, sizeof(VertexId), hipMemcpyDeviceToHost);
            hipMemcpy((void*)&temp_sigma, problem->data_slices[gpu]->sigmas.GetPointer(util::DEVICE)+child_, sizeof(Value   ), hipMemcpyDeviceToHost);
            hipMemcpy((void*)&temp_bc, problem->data_slices[gpu]->bc_values.GetPointer(util::DEVICE)+child_, sizeof(Value), hipMemcpyDeviceToHost);
            cout<<"\t "<<temp_label<<"\t "<<temp_sigma<<"\t "<<temp_bc;

            hipMemcpy((void*)&temp_label, problem->data_slices[gpu]->labels.GetPointer(util::DEVICE)+parent_, sizeof(VertexId), hipMemcpyDeviceToHost);
            hipMemcpy((void*)&temp_sigma, problem->data_slices[gpu]->sigmas.GetPointer(util::DEVICE)+parent_, sizeof(Value   ), hipMemcpyDeviceToHost);
            hipMemcpy((void*)&temp_bc, problem->data_slices[gpu]->bc_values.GetPointer(util::DEVICE)+parent_, sizeof(Value), hipMemcpyDeviceToHost);
            cout<<"\t "<<temp_label<<"\t "<<temp_sigma<<"\t "<<temp_bc<<endl;
        }
    }*/

    // Verify the result
    if (reference_check_bc_values != NULL)
    {
        //util::cpu_mt::PrintCPUArray<SizeT, Value>("reference_check_bc_values", reference_check_bc_values, graph->nodes);
        //util::cpu_mt::PrintCPUArray<SizeT, Value>("bc_values", h_bc_values, graph->nodes);
        if (!quiet_mode) { printf("Validity BC Value: "); }
        int num_error = CompareResults(
                            h_bc_values, reference_check_bc_values,
                            graph->nodes, true, quiet_mode);
        if (num_error > 0)
        {
            if (!quiet_mode) { printf("Number of errors occurred: %d\n", num_error); }
        }
        if (!quiet_mode) { printf("\n"); }
    }
    if (reference_check_ebc_values != NULL)
    {
        if (!quiet_mode) { printf("Validity Edge BC Value: "); }
        int num_error = CompareResults(
                            h_ebc_values, reference_check_ebc_values,
                            graph->edges, true, quiet_mode);
        if (num_error > 0)
        {
            if (!quiet_mode) { printf("Number of errors occurred: %d\n", num_error); }
        }
        if (!quiet_mode) { printf("\n"); }
    }
    if (reference_check_sigmas != NULL)
    {
        if (!quiet_mode) { printf("Validity Sigma: "); }
        int num_error = CompareResults(
                            h_sigmas, reference_check_sigmas,
                            graph->nodes, true, quiet_mode);
        if (num_error > 0)
        {
            if (!quiet_mode)
            {
                printf("Number of errors occurred: %d\n", num_error);
            }
        }
        if (!quiet_mode) { printf("\n"); }
    }
    if (reference_check_labels != NULL)
    {
        if (!quiet_mode) { printf("Validity labels: "); }
        int num_error = CompareResults(
                            h_labels, reference_check_labels,
                            graph->nodes, true, quiet_mode);
        if (num_error > 0)
        {
            if (!quiet_mode)
            {
                printf("Number of errors occurred: %d\n", num_error);
            }
        }
        if (!quiet_mode) { printf("\n"); }
    }

    if (!quiet_mode)
    {
        // Display Solution
        DisplaySolution(h_sigmas, h_bc_values, graph->nodes);
    }

    info->ComputeTraversalStats(  // compute running statistics
        enactor->enactor_stats.GetPointer(), elapsed, h_labels);

    if (!quiet_mode)
    {
        info->DisplayStats();  // display collected statistics
    }

    info->CollectInfo();  // collected all the info and put into JSON mObject

    if (!quiet_mode)
    {
        printf("\n\tMemory Usage(B)\t");
        for (int gpu = 0; gpu < num_gpus; gpu++)
            if (num_gpus > 1) {if (gpu != 0) printf(" #keys%d,0\t #keys%d,1\t #ins%d,0\t #ins%d,1", gpu, gpu, gpu, gpu); else printf(" #keys%d,0\t #keys%d,1", gpu, gpu);}
            else printf(" #keys%d,0\t #keys%d,1", gpu, gpu);
        if (num_gpus > 1) printf(" #keys%d", num_gpus);
        printf("\n");
        double max_queue_sizing_[2] = {0, 0}, max_in_sizing_ = 0;
        for (int gpu = 0; gpu < num_gpus; gpu++)
        {
            size_t gpu_free, dummy;
            hipSetDevice(gpu_idx[gpu]);
            hipMemGetInfo(&gpu_free, &dummy);
            printf("GPU_%d\t %ld", gpu_idx[gpu], org_size[gpu] - gpu_free);
            for (int i = 0; i < num_gpus; i++)
            {
                for (int j = 0; j < 2; j++)
                {
                    SizeT x = problem->data_slices[gpu]->frontier_queues[i].keys[j].GetSize();
                    printf("\t %lld", (long long) x);
                    double factor = 1.0 * x / (num_gpus > 1 ? problem->graph_slices[gpu]->in_counter[i] : problem->graph_slices[gpu]->nodes);
                    if (factor > max_queue_sizing_[j]) max_queue_sizing_[j] = factor;
                }
                if (num_gpus > 1 && i != 0 )
                    for (int t = 0; t < 2; t++)
                    {
                        SizeT x = problem->data_slices[gpu][0].keys_in[t][i].GetSize();
                        printf("\t %lld", (long long) x);
                        double factor = 1.0 * x / problem->graph_slices[gpu]->in_counter[i];
                        if (factor > max_in_sizing_) max_in_sizing_ = factor;
                    }
            }
            if (num_gpus > 1) printf("\t %lld", (long long)(problem->data_slices[gpu]->frontier_queues[num_gpus].keys[0].GetSize()));
            printf("\n");
        }
        printf("\t queue_sizing =\t %lf \t %lf", max_queue_sizing_[0], max_queue_sizing_[1]);
        if (num_gpus > 1) printf("\t in_sizing =\t %lf", max_in_sizing_);
        printf("\n");
    }

    // Cleanup
    if (org_size            ) {delete[] org_size            ; org_size             = NULL;}
    if (problem             ) {delete   problem             ; problem              = NULL;}
    if (enactor             ) {delete   enactor             ; enactor              = NULL;}
    if (reference_sigmas    ) {delete[] reference_sigmas    ; reference_sigmas     = NULL;}
    if (reference_bc_values ) {delete[] reference_bc_values ; reference_bc_values  = NULL;}
    if (reference_ebc_values) {delete[] reference_ebc_values; reference_ebc_values = NULL;}
    if (reference_labels    ) {delete[] reference_labels    ; reference_labels     = NULL;}
    if (h_sigmas            ) {delete[] h_sigmas            ; h_sigmas             = NULL;}
    if (h_bc_values         ) {delete[] h_bc_values         ; h_bc_values          = NULL;}
    if (h_ebc_values        ) {delete[] h_ebc_values        ; h_ebc_values         = NULL;}
    if (h_labels            ) {delete[] h_labels            ; h_labels             = NULL;}
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 * @tparam DEBUG
 *
 * @param[in] info Pointer to mObject info.
 */
template <
    typename      VertexId,
    typename      Value,
    typename      SizeT,
    bool          INSTRUMENT,
    bool          DEBUG >
void RunTests_size_check(Info<VertexId, Value, SizeT> *info)
{
    if (info->info["size_check"].get_bool())
    {
        RunTests<VertexId, Value, SizeT, INSTRUMENT, DEBUG,  true>(info);
    }
    else
    {
        RunTests<VertexId, Value, SizeT, INSTRUMENT, DEBUG, false>(info);
    }
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 *
 * @param[in] info Pointer to mObject info.
 */
template <
    typename    VertexId,
    typename    Value,
    typename    SizeT,
    bool        INSTRUMENT >
void RunTests_debug(Info<VertexId, Value, SizeT> *info)
{
    if (info->info["debug_mode"].get_bool())
    {
        RunTests_size_check<VertexId, Value, SizeT, INSTRUMENT,  true>(info);
    }
    else
    {
        RunTests_size_check<VertexId, Value, SizeT, INSTRUMENT, false>(info);
    }
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] info Pointer to mObject info.
 */
template <
    typename      VertexId,
    typename      Value,
    typename      SizeT >
void RunTests_instrumented(Info<VertexId, Value, SizeT> *info)
{
    if (info->info["instrument"].get_bool())
    {
        RunTests_debug<VertexId, Value, SizeT,  true>(info);
    }
    else
    {
        RunTests_debug<VertexId, Value, SizeT, false>(info);
    }
}

/******************************************************************************
 * Main
 ******************************************************************************/

int main(int argc, char** argv)
{
    CommandLineArgs args(argc, argv);
    int graph_args = argc - args.ParsedArgc() - 1;
    if (argc < 2 || graph_args < 1 || args.CheckCmdLineFlag("help"))
    {
        Usage();
        return 1;
    }

    typedef int VertexId;  // Use int as the vertex identifier
    typedef float Value;   // Use float as the value type
    typedef int SizeT;     // Use int as the graph size type

    Csr<VertexId, Value, SizeT> csr(false);  // graph we process on
    Info<VertexId, Value, SizeT> *info = new Info<VertexId, Value, SizeT>;

    // graph construction or generation related parameters
    info->info["undirected"] = true;   // require undirected input graph
    info->info["edge_value"] = false;  // don't need per edge weight values

    info->Init("BC", args, csr);  // initialize Info structure
    RunTests_instrumented<VertexId, Value, SizeT>(info);  // run test

    return 0;
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:

#include "hip/hip_runtime.h"
// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_wtf.cu
 *
 * @brief Simple test driver program for computing Pagerank.
 */

#include <stdio.h>
#include <string>
#include <deque>
#include <vector>
#include <iostream>
#include <cstdlib>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph construction utils
#include <gunrock/graphio/market.cuh>

// BFS includes
#include <gunrock/app/wtf/wtf_enactor.cuh>
#include <gunrock/app/wtf/wtf_problem.cuh>
#include <gunrock/app/wtf/wtf_functor.cuh>

// Operator includes
#include <gunrock/oprtr/advance/kernel.cuh>
#include <gunrock/oprtr/filter/kernel.cuh>

#include <moderngpu.cuh>

// boost includes
#include <boost/config.hpp>
#include <boost/utility.hpp>
#include <boost/graph/adjacency_list.hpp>
#include <boost/graph/page_rank.hpp>


using namespace gunrock;
using namespace gunrock::app;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::wtf;


/******************************************************************************
 * Defines, constants, globals
 ******************************************************************************/

//bool g_verbose;
//bool g_undirected;
//bool g_quick;
//bool g_stream_from_host;

template <typename VertexId, typename Value>
struct RankPair {
    VertexId        vertex_id;
    Value           page_rank;

    RankPair(VertexId vertex_id, Value page_rank) : vertex_id(vertex_id), page_rank(page_rank) {}
};

template<typename RankPair>
bool PRCompare(
    RankPair elem1,
    RankPair elem2)
{
    return elem1.page_rank > elem2.page_rank;
}

/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/
void Usage()
{
    printf("\ntest_wtf <graph type> <graph type args> [--device=<device_index>] "
           "[--undirected] [--instrumented] [--quick] "
           "[--v]\n"
           "\n"
           "Graph types and args:\n"
           "  market [<file>]\n"
           "    Reads a Matrix-Market coordinate-formatted graph of directed/undirected\n"
           "    edges from stdin (or from the optionally-specified file).\n"
           "  --device=<device_index>  Set GPU device for running the graph primitive.\n"
           "  --undirected If set then treat the graph as undirected.\n"
           "  --instrumented If set then kernels keep track of queue-search_depth\n"
           "  and barrier duty (a relative indicator of load imbalance.)\n"
           "  --quick If set will skip the CPU validation code.\n"
        );
}

/**
 * @brief Displays the BFS result (i.e., distance from source)
 *
 * @param[in] node_id Pointer to node ID array
 * @param[in] rank Pointer to node rank score array
 * @param[in] nodes Number of nodes in the graph.
 */
template<typename VertexId, typename Value, typename SizeT>
void DisplaySolution(VertexId *node_id, Value *rank, SizeT nodes)
{
    // Print out at most top 10 largest components
    int top = (nodes < 10) ? nodes : 10;
    printf("Top %d Page Ranks:\n", top);
    for (int i = 0; i < top; ++i)
    {
        printf("Vertex ID: %d, Page Rank: %5f\n", node_id[i], rank[i]);
    }
}

/******************************************************************************
 * WTF Testing Routines
 *****************************************************************************/

/**
 * @brief A simple CPU-based reference WTF implementation.
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] src Source node ID for WTF algorithm
 * @param[out] node_id Pointer to store computed output node ID
 * @param[in] rank Host-side vector to store CPU computed labels for each node
 * @param[in] delta Delta value for computing PageRank score
 * @param[in] alpha Parameter to adjust iteration number
 * @param[in] max_iter max iteration to go
 */
// TODO: Boost PageRank cannot handle personalized pagerank, so currently the CPU
// implementation gives incorrect answer. Need to find a CPU PPR implementation
template<
    typename VertexId,
    typename Value,
    typename SizeT>
void SimpleReferenceWTF(
    const Csr<VertexId, Value, SizeT>       &graph,
    VertexId                                src,
    VertexId                                *node_id,
    Value                                   *rank,
    Value                                   delta,
    Value                                   alpha,
    SizeT                                   max_iter)
{
    using namespace boost;

    //Preparation
    typedef adjacency_list<vecS, vecS, bidirectionalS, no_property,
                           property<edge_index_t, int> > Graph;

    Graph g;

    for (int i = 0; i < graph.nodes; ++i)
    {
        for (int j = graph.row_offsets[i]; j < graph.row_offsets[i+1]; ++j)
        {
            Graph::edge_descriptor e =
                add_edge(i, graph.column_indices[j], g).first;
            put(edge_index, g, e, i);
        }
    }


    //
    //compute page rank
    //

    CpuTimer cpu_timer;
    cpu_timer.Start();

    //remove_dangling_links(g);

    std::vector<Value> ranks(num_vertices(g));
    page_rank(g, make_iterator_property_map(
                  ranks.begin(), get(boost::vertex_index, g)),
              boost::graph::n_iterations(max_iter));

    cpu_timer.Stop();
    float elapsed = cpu_timer.ElapsedMillis();

    for (std::size_t i = 0; i < num_vertices(g); ++i)
    {
        rank[i] = ranks[i];
    }

    //sort the top page ranks
    RankPair<SizeT, Value> *pr_list =
        (RankPair<SizeT, Value>*)malloc(
            sizeof(RankPair<SizeT, Value>) * num_vertices(g));
    for (int i = 0; i < num_vertices(g); ++i)
    {
        pr_list[i].vertex_id = i;
        pr_list[i].page_rank = rank[i];
    }
    std::stable_sort(
        pr_list, pr_list + num_vertices(g), PRCompare<RankPair<SizeT, Value> >);

    std::vector<int> in_degree(num_vertices(g));
    std::vector<Value> refscore(num_vertices(g));

    for (int i = 0; i < num_vertices(g); ++i)
    {
        node_id[i] = pr_list[i].vertex_id;
        rank[i] = (i == src) ? 1.0 : 0;
        in_degree[i] = 0;
        refscore[i] = 0;
    }

    free(pr_list);

    int cot_size = (graph.nodes > 1000) ? 1000 : graph.nodes;

    for (int i = 0; i < cot_size; ++i)
    {
        int node = node_id[i];
        for (int j = graph.row_offsets[node];
             j < graph.row_offsets[node+1]; ++j)
        {
            VertexId edge = graph.column_indices[j];
            ++in_degree[edge];
        }
    }

    int salsa_iter = 1.0/alpha+1;
    for (int iter = 0; iter < salsa_iter; ++iter)
    {
        for (int i = 0; i < cot_size; ++i)
        {
            int node = node_id[i];
            int out_degree = graph.row_offsets[node+1]-graph.row_offsets[node];
            for (int j = graph.row_offsets[node];
                 j < graph.row_offsets[node+1]; ++j)
            {
                VertexId edge = graph.column_indices[j];
                Value val = rank[node]/ (out_degree > 0 ? out_degree : 1.0);
                refscore[edge] += val;
            }
        }
        for (int i = 0; i < cot_size; ++i)
        {
            rank[node_id[i]] = 0;
        }

        for (int i = 0; i < cot_size; ++i)
        {
            int node = node_id[i];
            rank[node] += (node == src) ? alpha : 0;
            for (int j = graph.row_offsets[node];
                 j < graph.row_offsets[node+1]; ++j)
            {
                VertexId edge = graph.column_indices[j];
                Value val = (1-alpha)*refscore[edge]/in_degree[edge];
                rank[node] += val;
            }
        }

        for (int i = 0; i < cot_size; ++i)
        {
            if (iter+1<salsa_iter) refscore[node_id[i]] = 0;
        }
    }

    //sort the top page ranks
    RankPair<SizeT, Value> *final_list =
        (RankPair<SizeT, Value>*)malloc(
            sizeof(RankPair<SizeT, Value>) * num_vertices(g));
    for (int i = 0; i < num_vertices(g); ++i)
    {
        final_list[i].vertex_id = node_id[i];
        final_list[i].page_rank = refscore[i];
    }
    std::stable_sort(
        final_list, final_list + num_vertices(g),
        PRCompare<RankPair<SizeT, Value> >);

    for (int i = 0; i < num_vertices(g); ++i)
    {
        node_id[i] = final_list[i].vertex_id;
        rank[i] = final_list[i].page_rank;
    }

    free(final_list);

    printf("CPU Who-To-Follow finished in %lf msec.\n", elapsed);
}

/**
 * @brief Run HITS tests
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] src Source node ID for WTF algorithm
 * @param[in] delta Delta value for computing WTF, usually set to .85
 * @param[in] alpha Parameter to adjust iteration number
 * @param[in] error Error threshold value
 * @param[in] max_iter Max iteration for WTF computing
 * @param[in] max_grid_size Maximum CTA occupancy
 * @param[in] num_gpus Number of GPUs
 * @param[in] context CudaContext for moderngpu to use
 *
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT,
    bool DEBUG,
    bool SIZE_CHECK>
void RunTests(Info<VertexId, Value, SizeT> *info)
{

    typedef WTFProblem<
        VertexId,
        SizeT,
        Value> Problem;

    Csr<VertexId, Value, SizeT>
                 *csr                 = info->csr_ptr;
    VertexId      src                   = info->info["source_vertex"].get_int64();
    int           max_grid_size         = info->info["max_grid_size"].get_int();
    int           num_gpus              = info->info["num_gpus"].get_int();
    bool          quick_mode            = info->info["quick_mode"].get_bool();
    bool          quiet_mode            = info->info["quiet_mode"].get_bool();
    bool          stream_from_host      = info->info["stream_from_host"].get_bool();
    Value         alpha                 = info->info["alpha"].get_real();
    Value         delta                 = info->info["delta"].get_real();
    Value         error                 = info->info["error"].get_real();
    SizeT         max_iter              = info->info["max_iteration"].get_int();
    ContextPtr    *context              = (ContextPtr*)info->context;

    json_spirit::mArray device_list = info->info["device_list"].get_array();
    int* gpu_idx = new int[num_gpus];
    for (int i = 0; i < num_gpus; i++) gpu_idx[i] = device_list[i].get_int();



    // Allocate host-side label array (for both reference and gpu-computed results)
    Value    *reference_rank    = (Value*)malloc(sizeof(Value) * csr->nodes);
    Value    *h_rank            = (Value*)malloc(sizeof(Value) * csr->nodes);
    VertexId *h_node_id         = (VertexId*)malloc(sizeof(VertexId) * csr->nodes);
    VertexId *reference_node_id = (VertexId*)malloc(sizeof(VertexId) * csr->nodes);
    Value    *reference_check   = (quick_mode) ? NULL : reference_rank;

    // Allocate WTF enactor map
    WTFEnactor<Problem, INSTRUMENT, DEBUG, SIZE_CHECK> wtf_enactor(gpu_idx);
    // Allocate problem on GPU
    Problem *problem = new Problem;
    util::GRError(problem->Init(
                      stream_from_host,
                      *csr,
                      num_gpus),
                  "Problem WTF Initialization Failed", __FILE__, __LINE__);

    // Perform WTF
    GpuTimer gpu_timer;

    util::GRError(
        problem->Reset(
            src, delta, alpha, error, wtf_enactor.GetFrontierType()),
        "pr Problem Data Reset Failed", __FILE__, __LINE__);
    gpu_timer.Start();
    util::GRError(
        wtf_enactor.template Enact<Problem>(
            *context, src, alpha, problem, max_iter, max_grid_size),
        "HITS Problem Enact Failed", __FILE__, __LINE__);
    gpu_timer.Stop();

    float elapsed = gpu_timer.ElapsedMillis();

    // Copy out results
    util::GRError(
        problem->Extract(h_rank, h_node_id),
        "HITS Problem Data Extraction Failed", __FILE__, __LINE__);

    float total_pr = 0;
    for (int i = 0; i < csr->nodes; ++i)
    {
        total_pr += h_rank[i];
    }

    //
    // Compute reference CPU HITS solution for source-distance
    //
    if (reference_check != NULL && total_pr > 0)
    {
        if (!quiet_mode) printf("compute ref value\n");
        SimpleReferenceWTF(
            *csr,
            src,
            reference_node_id,
            reference_check,
            delta,
            alpha,
            max_iter);
        if (!quiet_mode) printf("\n");
    }

    // Verify the result
    if (reference_check != NULL && total_pr > 0)
    {
        if (!quiet_mode) printf("Validity: ");
        CompareResults(h_rank, reference_check, csr->nodes, true);
    }

    if (!quiet_mode) {
        printf("\nGPU result.");
        DisplaySolution(h_node_id, h_rank, csr->nodes);
    }

    info->ComputeCommonStats(wtf_enactor.enactor_stats.GetPointer(), elapsed);

    if (!quiet_mode)
        info->DisplayStats();

    info->CollectInfo();

    // Cleanup
    if (problem) delete problem;
    if (reference_check) free(reference_check);
    if (h_rank) free(h_rank);

    hipDeviceSynchronize();
}

template <
    typename      VertexId,
    typename      Value,
    typename      SizeT,
    bool          INSTRUMENT,
    bool          DEBUG>
void RunTests_size_check(Info<VertexId, Value, SizeT> *info)
{
    if (info->info["size_check"].get_bool()) RunTests
        <VertexId, Value, SizeT, INSTRUMENT, DEBUG,
        true > (info);
   else RunTests
        <VertexId, Value, SizeT, INSTRUMENT, DEBUG,
        false> (info);
}

template <
    typename    VertexId,
    typename    Value,
    typename    SizeT,
    bool        INSTRUMENT>
void RunTests_debug(Info<VertexId, Value, SizeT> *info)
{
    if (info->info["debug_mode"].get_bool()) RunTests_size_check
        <VertexId, Value, SizeT, INSTRUMENT,
        true > (info);
    else RunTests_size_check
        <VertexId, Value, SizeT, INSTRUMENT,
        false> (info);
}

template <
    typename      VertexId,
    typename      Value,
    typename      SizeT>
void RunTests_instrumented(Info<VertexId, Value, SizeT> *info)
{
    if (info->info["instrument"].get_bool()) RunTests_debug
        <VertexId, Value, SizeT,
        true > (info);
    else RunTests_debug
        <VertexId, Value, SizeT,
        false> (info);
}


/******************************************************************************
 * Main
 ******************************************************************************/
int main( int argc, char** argv)
{
    CommandLineArgs args(argc, argv);
    int graph_args = argc - args.ParsedArgc() - 1;
    if ((argc < 2) || (args.CheckCmdLineFlag("help")))
    {
        Usage();
        return 1;
    }

    //
    // Construct graph and perform search(es)
    //
    typedef int VertexId;                   // Use as the node identifier
    typedef float Value;                    // Use as the value type
    typedef int SizeT;                      // Use as the graph size type
    Csr<VertexId, Value, SizeT> csr(false); // default for stream_from_host
    Info<VertexId, Value, SizeT> *info = new Info<VertexId, Value, SizeT>;

    info->info["undirected"] = args.CheckCmdLineFlag("undirected");

    info->Init("WTF", args, csr);
    RunTests_instrumented<VertexId, Value, SizeT>(info);
    
    return 0;
}

// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_sssp.cu
 *
 * @brief Simple test driver program for single source shorest path.
 */

#include <stdio.h>
#include <string>
#include <deque>
#include <vector>
#include <iostream>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// SSSP includes
#include <gunrock/app/sssp/sssp_enactor.cuh>
#include <gunrock/app/sssp/sssp_problem.cuh>
#include <gunrock/app/sssp/sssp_functor.cuh>

// Operator includes
#include <gunrock/oprtr/advance/kernel.cuh>
#include <gunrock/oprtr/filter/kernel.cuh>
#include <gunrock/priority_queue/kernel.cuh>

#include <moderngpu.cuh>

// Boost includes for CPU dijkstra SSSP reference algorithms
#include <boost/config.hpp>
#include <boost/graph/graph_traits.hpp>
#include <boost/graph/adjacency_list.hpp>
#include <boost/graph/dijkstra_shortest_paths.hpp>
#include <boost/property_map/property_map.hpp>

using namespace gunrock;
using namespace gunrock::app;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::sssp;

/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/
void Usage()
{
    printf(
        " test_sssp <graph type> <graph type args> [--device=<device_index>]\n"
        " [--undirected] [--instrumented] [--src=<source index>] [--quick=<0|1>]\n"
        " [--mark-pred] [--queue-sizing=<scale factor>] [--traversal-mode=<0|1>]\n"
        " [--in-sizing=<in/out queue scale factor>] [--disable-size-check]\n"
        " [--grid-size=<grid size>] [partition_method=<random|biasrandom|clustered|metis>]\n"
        " [--v] [--iteration-num=<num>]\n"
        "\n"
        "Graph types and args:\n"
        "  market [<file>]\n"
        "    Reads a Matrix-Market coordinate-formatted graph of directed / undirected\n"
        "    edges from stdin (or from the optionally-specified file).\n"
        "  --device=<device_index>   Set GPU device for running the test. [Default: 0].\n"
        "  --undirected              Treat the graph as undirected (symmetric).\n"
        "  --instrumented            Keep kernels statics [Default: Disable].\n"
        "                            total_queued, search_depth and barrier duty\n"
        "                            (a relative indicator of load imbalance.)\n"
        "  --src=<source vertex id>  Begins SSSP from the source [Default: 0].\n"
        "                            If randomize: from a random source vertex.\n"
        "                            If largestdegree: from largest degree vertex.\n"
        "  --quick=<0 or 1>          Skip the CPU validation: 1, or not: 0 [Default: 1].\n"
        "  --mark-pred               Keep both label info and predecessor info.\n"
        "  --queue-sizing=<factor>   Allocates a frontier queue sized at:\n"
        "                            (graph-edges * <scale factor>) [Default: 1.0].\n"
        "  --v                       Print verbose per iteration debug info.\n"
        "  --iteration-num=<number>  Number of runs to perform the test [Default: 1].\n"
        "  --traversal-mode=<0 or 1> Set traversal strategy, 0 for Load-Balanced,\n"
        "                            1 for Dynamic-Cooperative [Default: dynamic\n"
        "                            determine based on average degree].\n"
    );
}

/**
 * @brief Displays the SSSP result (i.e., distance from source)
 *
 * @tparam VertexId
 * @tparam SizeT
 *
 * @param[in] source_path Search depth from the source for each node.
 * @param[in] num_nodes Number of nodes in the graph.
 */
template<typename VertexId, typename SizeT>
void DisplaySolution (VertexId *source_path, SizeT num_nodes)
{
    if (num_nodes > 40) num_nodes = 40;

    printf("[");
    for (VertexId i = 0; i < num_nodes; ++i)
    {
        PrintValue(i);
        printf(":");
        PrintValue(source_path[i]);
        printf(" ");
    }
    printf("]\n");
}

/******************************************************************************
 * SSSP Testing Routines
 *****************************************************************************/

/**
 * @brief A simple CPU-based reference SSSP ranking implementation.
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam MARK_PREDECESSORS
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] node_values Host-side vector to store CPU computed labels for each node
 * @param[in] node_preds Host-side vector to store CPU computed predecessors for each node
 * @param[in] src Source node where SSSP starts
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool     MARK_PREDECESSORS >
void SimpleReferenceSssp(
    const Csr<VertexId, Value, SizeT> &graph,
    Value                             *node_values,
    VertexId                          *node_preds,
    VertexId                          src,
    bool                              quiet)
{
    using namespace boost;

    // Prepare Boost Datatype and Data structure
    typedef adjacency_list<vecS, vecS, directedS, no_property,
            property <edge_weight_t, unsigned int> > Graph;

    typedef graph_traits<Graph>::vertex_descriptor vertex_descriptor;
    typedef graph_traits<Graph>::edge_descriptor edge_descriptor;

    typedef std::pair<VertexId, VertexId> Edge;

    Edge   *edges = ( Edge*)malloc(sizeof( Edge) * graph.edges);
    Value *weight = (Value*)malloc(sizeof(Value) * graph.edges);

    for (int i = 0; i < graph.nodes; ++i)
    {
        for (int j = graph.row_offsets[i]; j < graph.row_offsets[i + 1]; ++j)
        {
            edges[j] = Edge(i, graph.column_indices[j]);
            weight[j] = graph.edge_values[j];
        }
    }

    Graph g(edges, edges + graph.edges, weight, graph.nodes);

    std::vector<Value> d(graph.nodes);
    std::vector<vertex_descriptor> p(graph.nodes);
    vertex_descriptor s = vertex(src, g);

    property_map<Graph, vertex_index_t>::type indexmap = get(vertex_index, g);

    //
    // Perform SSSP
    //

    CpuTimer cpu_timer;
    cpu_timer.Start();

    if (MARK_PREDECESSORS)
    {
        dijkstra_shortest_paths(g, s,
                                predecessor_map(boost::make_iterator_property_map(
                                        p.begin(), get(boost::vertex_index, g))).distance_map(
                                    boost::make_iterator_property_map(
                                        d.begin(), get(boost::vertex_index, g))));
    }
    else
    {
        dijkstra_shortest_paths(g, s,
                                distance_map(boost::make_iterator_property_map(
                                        d.begin(), get(boost::vertex_index, g))));
    }
    cpu_timer.Stop();
    float elapsed = cpu_timer.ElapsedMillis();

    if (!quiet) { printf("CPU SSSP finished in %lf msec.\n", elapsed); }

    Coo<Value, Value>* sort_dist = NULL;
    Coo<VertexId, VertexId>* sort_pred = NULL;
    sort_dist = (Coo<Value, Value>*)malloc(
                    sizeof(Coo<Value, Value>) * graph.nodes);
    if (MARK_PREDECESSORS)
    {
        sort_pred = (Coo<VertexId, VertexId>*)malloc(
                        sizeof(Coo<VertexId, VertexId>) * graph.nodes);
    }
    graph_traits < Graph >::vertex_iterator vi, vend;
    for (tie(vi, vend) = vertices(g); vi != vend; ++vi)
    {
        sort_dist[(*vi)].row = (*vi);
        sort_dist[(*vi)].col = d[(*vi)];
    }
    std::stable_sort(
        sort_dist, sort_dist + graph.nodes,
        RowFirstTupleCompare<Coo<Value, Value> >);

    if (MARK_PREDECESSORS)
    {
        for (tie(vi, vend) = vertices(g); vi != vend; ++vi)
        {
            sort_pred[(*vi)].row = (*vi);
            sort_pred[(*vi)].col = p[(*vi)];
        }
        std::stable_sort(
            sort_pred, sort_pred + graph.nodes,
            RowFirstTupleCompare< Coo<VertexId, VertexId> >);
    }

    for (int i = 0; i < graph.nodes; ++i)
    {
        node_values[i] = sort_dist[i].col;
    }
    if (MARK_PREDECESSORS)
    {
        for (int i = 0; i < graph.nodes; ++i)
        {
            node_preds[i] = sort_pred[i].col;
        }
    }
    if (sort_dist) free(sort_dist);
    if (sort_pred) free(sort_pred);
}


/**
 * @brief Run SSSP tests
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 * @tparam MARK_PREDECESSORS
 *
 * @param[in] parameter Pointer to test parameter settings
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT,
    bool DEBUG,
    bool SIZE_CHECK,
    bool MARK_PREDECESSORS >
void RunTests(Info<VertexId, Value, SizeT> *info)
{
    typedef SSSPProblem < VertexId,
            SizeT,
            Value,
            MARK_PREDECESSORS > Problem;

    typedef SSSPEnactor < Problem,
            INSTRUMENT,
            DEBUG,
            SIZE_CHECK > Enactor;

    // parse configurations from mObject info
    Csr<VertexId, Value, SizeT> *graph = info->csr_ptr;
    VertexId src                 = info->info["source_vertex"].get_int64();
    int max_grid_size            = info->info["max_grid_size"].get_int();
    int num_gpus                 = info->info["num_gpus"].get_int();
    double max_queue_sizing      = info->info["max_queue_sizing"].get_real();
    double max_queue_sizing1     = info->info["max_queue_sizing1"].get_real();
    double max_in_sizing         = info->info["max_in_sizing"].get_real();
    std::string partition_method = info->info["partition_method"].get_str();
    double partition_factor      = info->info["partition_factor"].get_real();
    int partition_seed           = info->info["partition_seed"].get_int();
    bool quiet_mode              = info->info["quiet_mode"].get_bool();
    bool quick_mode              = info->info["quick_mode"].get_bool();
    bool stream_from_host        = info->info["stream_from_host"].get_bool();
    int traversal_mode           = info->info["traversal_mode"].get_int();
    int iterations               = info->info["num_iteration"].get_int();
    int delta_factor             = info->info["delta_factor"].get_int();

    json_spirit::mArray device_list = info->info["device_list"].get_array();
    int* gpu_idx = new int[num_gpus];
    for (int i = 0; i < num_gpus; i++) gpu_idx[i] = device_list[i].get_int();

    // TODO: remove after merge mgpu-cq
    ContextPtr   *context = (ContextPtr*)  info->context;
    hipStream_t *streams = (hipStream_t*)info->streams;

    // Allocate host-side array (for both reference and GPU-computed results)
    Value    *reference_labels      = new Value[graph->nodes];
    Value    *h_labels              = new Value[graph->nodes];
    Value    *reference_check_label = (quick_mode) ? NULL : reference_labels;
    VertexId *reference_preds       = MARK_PREDECESSORS ? new VertexId[graph->nodes] : NULL;
    VertexId *h_preds               = MARK_PREDECESSORS ? new VertexId[graph->nodes] : NULL;
    VertexId *reference_check_pred  = (quick_mode || !MARK_PREDECESSORS) ? NULL : reference_preds;

    size_t *org_size = new size_t[num_gpus];
    for (int gpu = 0; gpu < num_gpus; gpu++)
    {
        size_t dummy;
        hipSetDevice(gpu_idx[gpu]);
        hipMemGetInfo(&(org_size[gpu]), &dummy);
    }

    // Allocate SSSP enactor map
    Enactor* enactor = new Enactor(num_gpus, gpu_idx);

    // Allocate problem on GPU
    Problem *problem = new Problem;
    util::GRError(problem->Init(
                      stream_from_host,
                      graph,
                      NULL,
                      num_gpus,
                      gpu_idx,
                      partition_method,
                      streams,
                      delta_factor,
                      max_queue_sizing,
                      max_in_sizing,
                      partition_factor,
                      partition_seed),
                  "SSSP Problem Init failed", __FILE__, __LINE__);
    util::GRError(enactor->Init(
        context, problem, max_grid_size, traversal_mode),
                  "SSSP Enactor Init failed", __FILE__, __LINE__);

    // compute reference CPU SSSP solution for source-distance
    if (reference_check_label != NULL)
    {
        if (!quiet_mode) { printf("Computing reference value ...\n"); }
        SimpleReferenceSssp<VertexId, Value, SizeT, MARK_PREDECESSORS>(
            *graph,
            reference_check_label,
            reference_check_pred,
            src,
            quiet_mode);
        if (!quiet_mode) { printf("\n"); }
    }

    double elapsed = 0.0f;

    // perform SSSP
    CpuTimer cpu_timer;

    for (int iter = 0; iter < iterations; ++iter)
    {
        util::GRError(problem->Reset(
                          src, enactor->GetFrontierType(), max_queue_sizing),
                      "SSSP Problem Data Reset Failed", __FILE__, __LINE__);
        util::GRError(enactor->Reset(),
                      "SSSP Enactor Reset failed", __FILE__, __LINE__);

        if (!quiet_mode)
        {
            printf("__________________________\n"); fflush(stdout);
        }
        cpu_timer.Start();
        util::GRError(enactor->Enact(src, traversal_mode),
                      "SSSP Problem Enact Failed", __FILE__, __LINE__);
        cpu_timer.Stop();
        if (!quiet_mode)
        {
            printf("--------------------------\n"); fflush(stdout);
        }
        elapsed += cpu_timer.ElapsedMillis();
    }
    elapsed /= iterations;

    // Copy out results
    util::GRError(problem->Extract(h_labels, h_preds),
                  "SSSP Problem Data Extraction Failed", __FILE__, __LINE__);

    for (SizeT i = 0; i < graph->nodes; i++)
    {
        if (reference_check_label[i] == -1)
        {
            reference_check_label[i] = util::MaxValue<Value>();
        }
    }

    if (!quiet_mode)
    {
        // Display Solution
        printf("\nFirst 40 labels of the GPU result.\n");
        DisplaySolution(h_labels, graph->nodes);
    }
    // Verify the result
    if (reference_check_label != NULL)
    {
        if (!quiet_mode) { printf("Label Validity: "); }
        int error_num = CompareResults(
                            h_labels, reference_check_label,
                            graph->nodes, true, quiet_mode);
        if (error_num > 0)
        {
            if (!quiet_mode) { printf("%d errors occurred.\n", error_num); }
        }
        if (!quiet_mode)
        {
            printf("\nFirst 40 labels of the reference CPU result.\n");
            DisplaySolution(reference_check_label, graph->nodes);
        }
    }

    info->ComputeTraversalStats(  // compute running statistics
        enactor->enactor_stats.GetPointer(), elapsed, h_labels);

    if (!quiet_mode)
    {
        info->DisplayStats();  // display collected statistics
    }

    info->CollectInfo();  // collected all the info and put into JSON mObject

    if (!quiet_mode)
    {
        if (MARK_PREDECESSORS)
        {
            printf("\nFirst 40 preds of the GPU result.\n");
            DisplaySolution(h_preds, graph->nodes);
            if (reference_check_label != NULL)
            {
                printf("\nFirst 40 preds of the reference CPU result (could be different because the paths are not unique).\n");
                DisplaySolution(reference_check_pred, graph->nodes);
            }
        }

        printf("\n\tMemory Usage(B)\t");
        for (int gpu = 0; gpu < num_gpus; gpu++)
            if (num_gpus > 1) {if (gpu != 0) printf(" #keys%d,0\t #keys%d,1\t #ins%d,0\t #ins%d,1", gpu, gpu, gpu, gpu); else printf(" #keys%d,0\t #keys%d,1", gpu, gpu);}
            else printf(" #keys%d,0\t #keys%d,1", gpu, gpu);
        if (num_gpus > 1) printf(" #keys%d", num_gpus);
        printf("\n");
        double max_queue_sizing_[2] = {0, 0}, max_in_sizing_ = 0;
        for (int gpu = 0; gpu < num_gpus; gpu++)
        {
            size_t gpu_free, dummy;
            hipSetDevice(gpu_idx[gpu]);
            hipMemGetInfo(&gpu_free, &dummy);
            printf("GPU_%d\t %ld", gpu_idx[gpu], org_size[gpu] - gpu_free);
            for (int i = 0; i < num_gpus; i++)
            {
                for (int j = 0; j < 2; j++)
                {
                    SizeT x = problem->data_slices[gpu]->frontier_queues[i].keys[j].GetSize();
                    printf("\t %lld", (long long) x);
                    double factor = 1.0 * x / (num_gpus > 1 ? problem->graph_slices[gpu]->in_counter[i] : problem->graph_slices[gpu]->nodes);
                    if (factor > max_queue_sizing_[j]) max_queue_sizing_[j] = factor;
                }
                if (num_gpus > 1 && i != 0 )
                    for (int t = 0; t < 2; t++)
                    {
                        SizeT x = problem->data_slices[gpu][0].keys_in[t][i].GetSize();
                        printf("\t %lld", (long long) x);
                        double factor = 1.0 * x / problem->graph_slices[gpu]->in_counter[i];
                        if (factor > max_in_sizing_) max_in_sizing_ = factor;
                    }
            }
            if (num_gpus > 1) printf("\t %lld", (long long)(problem->data_slices[gpu]->frontier_queues[num_gpus].keys[0].GetSize()));
            printf("\n");
        }
        printf("\t queue_sizing =\t %lf \t %lf", max_queue_sizing_[0], max_queue_sizing_[1]);
        if (num_gpus > 1) printf("\t in_sizing =\t %lf", max_in_sizing_);
        printf("\n");
    }

    // Clean up
    if (org_size        ) {delete[] org_size        ; org_size         = NULL;}
    if (enactor         ) {delete   enactor         ; enactor          = NULL;}
    if (problem         ) {delete   problem         ; problem          = NULL;}
    if (reference_labels) {delete[] reference_labels; reference_labels = NULL;}
    if (h_labels        ) {delete[] h_labels        ; h_labels         = NULL;}
    if (reference_preds ) {delete[] reference_preds ; reference_preds  = NULL;}
    if (h_preds         ) {delete[] h_preds         ; h_preds          = NULL;}
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 * @tparam DEBUG
 * @tparam SIZE_CHECK
 *
 * @param[in] info Pointer to mObject info.
 */
template <
    typename    VertexId,
    typename    Value,
    typename    SizeT,
    bool        INSTRUMENT,
    bool        DEBUG,
    bool        SIZE_CHECK >
void RunTests_mark_predecessors(Info<VertexId, Value, SizeT> *info)
{
    if (info->info["mark_predecessors"].get_bool())
    {
        RunTests<VertexId, Value, SizeT, INSTRUMENT,
                 DEBUG, SIZE_CHECK, true>(info);
    }
    else
    {
        RunTests<VertexId, Value, SizeT, INSTRUMENT,
                 DEBUG, SIZE_CHECK, false>(info);
    }
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 * @tparam DEBUG
 *
 * @param[in] info Pointer to mObject info.
 */
template <
    typename      VertexId,
    typename      Value,
    typename      SizeT,
    bool          INSTRUMENT,
    bool          DEBUG >
void RunTests_size_check(Info<VertexId, Value, SizeT> *info)
{
    if (info->info["size_check"].get_bool())
    {
        RunTests_mark_predecessors<VertexId, Value, SizeT, INSTRUMENT,
                                   DEBUG,  true>(info);
    }
    else
    {
        RunTests_mark_predecessors<VertexId, Value, SizeT, INSTRUMENT,
                                   DEBUG, false>(info);
    }
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 *
 * @param[in] info Pointer to mObject info.
 */
template <
    typename    VertexId,
    typename    Value,
    typename    SizeT,
    bool        INSTRUMENT >
void RunTests_debug(Info<VertexId, Value, SizeT> *info)
{
    if (info->info["debug_mode"].get_bool())
    {
        RunTests_size_check<VertexId, Value, SizeT, INSTRUMENT,  true>(info);
    }
    else
    {
        RunTests_size_check<VertexId, Value, SizeT, INSTRUMENT, false>(info);
    }
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] info Pointer to mObject info.
 */
template <
    typename      VertexId,
    typename      Value,
    typename      SizeT >
void RunTests_instrumented(Info<VertexId, Value, SizeT> *info)
{
    if (info->info["instrument"].get_bool())
    {
        RunTests_debug<VertexId, Value, SizeT, true>(info);
    }
    else
    {
        RunTests_debug<VertexId, Value, SizeT, false>(info);
    }
}

/******************************************************************************
* Main
******************************************************************************/

int main(int argc, char** argv)
{
    CommandLineArgs args(argc, argv);
    int graph_args = argc - args.ParsedArgc() - 1;
    if (argc < 2 || graph_args < 1 || args.CheckCmdLineFlag("help"))
    {
        Usage();
        return 1;
    }

    typedef int VertexId;  // Use int as the vertex identifier
    typedef int Value;     // Use int as the value type
    typedef int SizeT;     // Use int as the graph size type

    Csr<VertexId, Value, SizeT> csr(false);  // graph we process on
    Info<VertexId, Value, SizeT> *info = new Info<VertexId, Value, SizeT>;

    // graph construction or generation related parameters
    info->info["undirected"] = args.CheckCmdLineFlag("undirected");
    info->info["edge_value"] = true;  // require per edge weight values

    info->Init("SSSP", args, csr);  // initialize Info structure
    RunTests_instrumented<VertexId, Value, SizeT>(info);  // run test

    return 0;
}

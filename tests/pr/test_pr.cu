#include "hip/hip_runtime.h"
// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_pr.cu
 *
 * @brief Simple test driver program for computing Pagerank.
 */

#include <stdio.h>
#include <string>
#include <deque>
#include <vector>
#include <iostream>
#include <cstdlib>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph construction utils
#include <gunrock/graphio/market.cuh>
#include <gunrock/graphio/rmat.cuh>
#include <gunrock/graphio/rgg.cuh>

// BFS includes
#include <gunrock/app/pr/pr_enactor.cuh>
#include <gunrock/app/pr/pr_problem.cuh>
#include <gunrock/app/pr/pr_functor.cuh>

// Operator includes
#include <gunrock/oprtr/advance/kernel.cuh>
#include <gunrock/oprtr/filter/kernel.cuh>

#include <moderngpu.cuh>

// boost includes
#include <boost/config.hpp>
#include <boost/utility.hpp>
#include <boost/graph/adjacency_list.hpp>
#include <boost/graph/page_rank.hpp>

using namespace gunrock;
using namespace gunrock::app;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::pr;


/******************************************************************************
 * Defines, constants, globals
 ******************************************************************************/

template <typename VertexId, typename Value>
struct RankPair
{
    VertexId        vertex_id;
    Value           page_rank;

    RankPair(VertexId vertex_id, Value page_rank) :
        vertex_id(vertex_id), page_rank(page_rank) {}
};

template<typename RankPair>
bool PRCompare(
    RankPair elem1,
    RankPair elem2)
{
    return elem1.page_rank > elem2.page_rank;
}

/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/

void Usage()
{
    printf(
        "\ntest_pr <graph type> <graph type args> [--device=<device_index>] "
        "[--undirected] [--instrumented] [--quick=<0|1>] [--v]\n"
        "\n"
        "Graph types and args:\n"
        "  market [<file>]\n"
        "    Reads a Matrix-Market coordinate-formatted graph of directed/undirected\n"
        "    edges from stdin (or from the optionally-specified file).\n"
        "  --device=<device_index>  Set GPU device for running the graph primitive.\n"
        "  --undirected If set then treat the graph as undirected.\n"
        "  --instrumented If set then kernels keep track of queue-search_depth\n"
        "  and barrier duty (a relative indicator of load imbalance.)\n"
        "  --quick If set will skip the CPU validation code. Default: 0\n"
    );
}

/**
 * @brief Displays the PageRank result
 *
 * @param[in] node_id Node vertex Id
 * @param[in] rank Rank value for the node
 * @param[in] nodes Number of nodes in the graph.
 */
template<typename VertexId, typename Value, typename SizeT>
void DisplaySolution(VertexId *node, Value *rank, SizeT nodes)
{
    int top = (nodes < 10) ? nodes : 10;  // at most top 10 ranked nodes
    printf("\nTop %d Ranked Vertices and PageRanks:\n", top);
    for (int i = 0; i < top; ++i)
    {
        printf("Vertex ID: %d, PageRank: %5f\n", node[i], rank[i]);
    }
}

/**
 * @brief Compares the equivalence of two arrays. If incorrect, print the location
 * of the first incorrect value appears, the incorrect value, and the reference
 * value.
 *
 * @tparam T datatype of the values being compared with.
 * @tparam SizeT datatype of the array length.
 *
 * @param[in] computed Vector of values to be compared.
 * @param[in] reference Vector of reference values
 * @param[in] len Vector length
 * @param[in] verbose Whether to print values around the incorrect one.
 *
 * \return Zero if two vectors are exactly the same, non-zero if there is any difference.
 */
template <typename SizeT>
int CompareResults_(
    float* computed,
    float* reference,
    SizeT len,
    bool verbose = true,
    bool quiet = false)
{
    float THRESHOLD = 0.05f;
    int flag = 0;
    for (SizeT i = 0; i < len; i++)
    {

        // Use relative error rate here.
        bool is_right = true;
        if (fabs(computed[i]) < 0.01f && fabs(reference[i] - 1) < 0.01f) continue;
        if (fabs(computed[i] - 0.0) < 0.01f)
        {
            if (fabs(computed[i] - reference[i]) > THRESHOLD)
                is_right = false;
        }
        else
        {
            if (fabs((computed[i] - reference[i]) / reference[i]) > THRESHOLD)
                is_right = false;
        }
        if (!is_right && flag == 0)
        {
            if (!quiet)
            {
                printf("\nINCORRECT: [%lu]: ", (unsigned long) i);
                PrintValue<float>(computed[i]);
                PrintValue<float>(reference[i]);

                if (verbose)
                {
                    printf("\nresult[...");
                    for (size_t j = (i >= 5) ? i - 5 : 0; (j < i + 5) && (j < len); j++)
                    {
                        PrintValue<float>(computed[j]);
                        printf(", ");
                    }
                    printf("...]");
                    printf("\nreference[...");
                    for (size_t j = (i >= 5) ? i - 5 : 0; (j < i + 5) && (j < len); j++)
                    {
                        PrintValue<float>(reference[j]);
                        printf(", ");
                    }
                    printf("...]");
                }
            }
            flag += 1;
        }
        if (!is_right && flag > 0) flag += 1;
    }
    if (!quiet)
    {
        printf("\n");
        if (!flag)
        {
            printf("CORRECT");
        }
    }
    return flag;
}

/******************************************************************************
 * PageRank Testing Routines
 *****************************************************************************/

/**
 * @brief A simple CPU-based reference Page Rank implementation.
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] node_id Source node for personalized PageRank (if any)
 * @param[in] rank Host-side vector to store CPU computed labels for each node
 * @param[in] delta delta for computing PR
 * @param[in] error error threshold
 * @param[in] max_iteration max iteration to go
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT >
void SimpleReferencePageRank(
    const Csr<VertexId, Value, SizeT> &graph,
    VertexId                          *node_id,
    Value                             *rank,
    Value                             delta,
    Value                             error,
    SizeT                             max_iteration,
    bool                              directed,
    bool                              quiet = false)
{
    using namespace boost;

    // preparation
    typedef adjacency_list< vecS, vecS, bidirectionalS, no_property,
            property<edge_index_t, int> > Graph;

    Graph g;

    for (int i = 0; i < graph.nodes; ++i)
    {
        for (int j = graph.row_offsets[i]; j < graph.row_offsets[i + 1]; ++j)
        {
            Graph::edge_descriptor e =
                add_edge(i, graph.column_indices[j], g).first;
            put(edge_index, g, e, i);
        }
    }

    // compute PageRank
    CpuTimer cpu_timer;
    cpu_timer.Start();

    std::vector<Value> ranks(num_vertices(g));
    page_rank(g, make_iterator_property_map(
                  ranks.begin(),
                  get(boost::vertex_index, g)),
              boost::graph::n_iterations(max_iteration));

    cpu_timer.Stop();
    float elapsed = cpu_timer.ElapsedMillis();

    for (std::size_t i = 0; i < num_vertices(g); ++i)
    {
        rank[i] = ranks[i];
    }

    // Sort the top ranked vertices
    RankPair<SizeT, Value> *pr_list =
        (RankPair<SizeT, Value>*)malloc(
            sizeof(RankPair<SizeT, Value>) * num_vertices(g));
    for (int i = 0; i < num_vertices(g); ++i)
    {
        pr_list[i].vertex_id = i;
        pr_list[i].page_rank = rank[i];
    }
    std::stable_sort(pr_list, pr_list + num_vertices(g),
                     PRCompare<RankPair<SizeT, Value> >);

    for (int i = 0; i < num_vertices(g); ++i)
    {
        node_id[i] = pr_list[i].vertex_id;
        rank[i] = pr_list[i].page_rank;
    }

    free(pr_list);
    if (!quiet) { printf("CPU PageRank finished in %lf msec.\n", elapsed); }
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 * @tparam DEBUG
 * @tparam SIZE_CHECK
 *
 * @param[in] parameter Pointer to test parameter settings
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT,
    bool DEBUG,
    bool SIZE_CHECK >
void RunTests(Info<VertexId, Value, SizeT> *info)
{
    typedef PRProblem <VertexId,
            SizeT,
            Value > PrProblem;

    typedef PREnactor <PrProblem,
            INSTRUMENT,
            DEBUG,
            SIZE_CHECK > PrEnactor;

    // parse configurations from mObject info
    Csr<VertexId, Value, SizeT> *graph = info->csr_ptr;
    VertexId src                 = info->info["source_vertex"].get_int64();
    bool undirected              = info->info["undirected"].get_bool();
    bool quiet_mode              = info->info["quiet_mode"].get_bool();
    bool quick_mode              = info->info["quick_mode"].get_bool();
    bool stream_from_host        = info->info["stream_from_host"].get_bool();
    int max_grid_size            = info->info["max_grid_size"].get_int();
    int num_gpus                 = info->info["num_gpus"].get_int();
    int max_iteration            = info->info["max_iteration"].get_int();
    double max_queue_sizing      = info->info["max_queue_sizing"].get_real();
    double max_queue_sizing1     = info->info["max_queue_sizing1"].get_real();
    double max_in_sizing         = info->info["max_in_sizing"].get_real();
    std::string partition_method = info->info["partition_method"].get_str();
    double partition_factor      = info->info["partition_factor"].get_real();
    int partition_seed           = info->info["partition_seed"].get_int();
    int iterations               = info->info["num_iteration"].get_int();
    int traversal_mode           = info->info["traversal_mode"].get_int();
    std::string ref_filename     = info->info["ref_filename"].get_str();
    Value delta                  = info->info["delta"].get_real();
    Value error                  = info->info["error"].get_real();

    json_spirit::mArray device_list = info->info["device_list"].get_array();
    int* gpu_idx = new int[num_gpus];
    for (int i = 0; i < num_gpus; i++) gpu_idx[i] = device_list[i].get_int();

    // TODO: remove after merge mgpu-cq
    ContextPtr   *context = (ContextPtr*)  info->context;
    hipStream_t *streams = (hipStream_t*)info->streams;

    // Allocate host-side array (for both reference and GPU-computed results)
    Value        *ref_rank           = new Value   [graph->nodes];
    Value        *h_rank             = new Value   [graph->nodes];
    VertexId     *h_node_id          = new VertexId[graph->nodes];
    VertexId     *ref_node_id        = new VertexId[graph->nodes];
    Value        *ref_check          = (quick_mode) ? NULL : ref_rank;

    size_t *org_size = new size_t[num_gpus];
    for (int gpu = 0; gpu < num_gpus; gpu++)
    {
        size_t dummy;
        hipSetDevice(gpu_idx[gpu]);
        hipMemGetInfo(&(org_size[gpu]), &dummy);
    }

    PrEnactor* enactor = new PrEnactor(num_gpus, gpu_idx);  // enactor map
    PrProblem *problem = new PrProblem;  // allocate problem on GPU

    util::GRError(problem->Init(
                      stream_from_host,
                      graph,
                      NULL,
                      num_gpus,
                      gpu_idx,
                      partition_method,
                      streams,
                      max_queue_sizing,
                      max_in_sizing,
                      partition_factor,
                      partition_seed),
                  "PR Problem Init failed", __FILE__, __LINE__);
    util::GRError(enactor->Init(
                      context, problem, traversal_mode, max_grid_size),
                  "PR Enactor Init failed", __FILE__, __LINE__);

    double elapsed = 0.0f;

    // perform PageRank
    CpuTimer cpu_timer;

    for (int iter = 0; iter < iterations; ++iter)
    {
        util::GRError(problem->Reset(
                          src, delta, error, max_iteration,
                          enactor->GetFrontierType(), max_queue_sizing),
                      "PR Problem Data Reset Failed", __FILE__, __LINE__);
        util::GRError(enactor->Reset(),
                      "PR Enactor Reset Reset failed", __FILE__, __LINE__);

        if (!quiet_mode)
        {
            printf("__________________________\n"); fflush(stdout);
        }
        cpu_timer.Start();
        util::GRError(enactor->Enact(traversal_mode),
                      "PR Problem Enact Failed", __FILE__, __LINE__);
        cpu_timer.Stop();
        if (!quiet_mode)
        {
            printf("--------------------------\n"); fflush(stdout);
        }
        elapsed += cpu_timer.ElapsedMillis();
    }
    elapsed /= iterations;

    // copy out results
    util::GRError(problem->Extract(h_rank, h_node_id),
                  "PR Problem Data Extraction Failed", __FILE__, __LINE__);

    if (!quiet_mode)
    {
        float total_pr = 0;
        for (int i = 0; i < graph->nodes; ++i)
        {
            total_pr += h_rank[i];
        }
        printf("Total rank : %f\n", total_pr);
    }

    // compute reference CPU solution
    if (ref_check != NULL)
    {
        if (!quiet_mode) { printf("Computing reference value ...\n"); }
        SimpleReferencePageRank <VertexId, Value, SizeT>(
            *graph,
            ref_node_id,
            ref_check,
            delta,
            error,
            max_iteration,
            !undirected,
            quiet_mode);
        if (!quiet_mode) { printf("\n"); }
    }

    // Verify the result
    if (ref_check != NULL)
    {
        if (!quiet_mode) { printf("Validity Rank: "); }
        int errors_count = CompareResults_(
                               h_rank, ref_check,
                               graph->nodes, true, quiet_mode);
        if (errors_count > 0)
        {
            if (!quiet_mode)
            {
                printf("number of errors : %lld\n", (long long) errors_count);
            }
        }
    }

    if (!quiet_mode)
    {
        printf("\nFirst 40 labels of the GPU result.");
        // Display Solution
        DisplaySolution(h_node_id, h_rank, graph->nodes);
    }

    info->ComputeCommonStats(  // compute running statistics
        enactor->enactor_stats.GetPointer(), elapsed);

    if (!quiet_mode)
    {
        info->DisplayStats();  // display collected statistics
    }

    info->CollectInfo();  // collected all the info and put into JSON mObject

    if (!quiet_mode)
    {
        printf("\n\tMemory Usage(B)\t");
        for (int gpu = 0; gpu < num_gpus; gpu++)
            if (num_gpus > 1) {if (gpu != 0) printf(" #keys%d,0\t #keys%d,1\t #ins%d,0\t #ins%d,1", gpu, gpu, gpu, gpu); else printf(" #keys%d,0\t #keys%d,1", gpu, gpu);}
            else printf(" #keys%d,0\t #keys%d,1", gpu, gpu);
        if (num_gpus > 1) printf(" #keys%d", num_gpus);
        printf("\n");
        double max_queue_sizing_[2] = {0, 0}, max_in_sizing_ = 0;
        for (int gpu = 0; gpu < num_gpus; gpu++)
        {
            size_t gpu_free, dummy;
            hipSetDevice(gpu_idx[gpu]);
            hipMemGetInfo(&gpu_free, &dummy);
            printf("GPU_%d\t %ld", gpu_idx[gpu], org_size[gpu] - gpu_free);
            for (int i = 0; i < num_gpus; i++)
            {
                for (int j = 0; j < 2; j++)
                {
                    SizeT x = problem->data_slices[gpu]->frontier_queues[i].keys[j].GetSize();
                    printf("\t %lld", (long long) x);
                    double factor = 1.0 * x / (num_gpus > 1 ? problem->graph_slices[gpu]->in_counter[i] : problem->graph_slices[gpu]->nodes);
                    if (factor > max_queue_sizing_[j]) max_queue_sizing_[j] = factor;
                }
                if (num_gpus > 1 && i != 0 )
                    for (int t = 0; t < 2; t++)
                    {
                        SizeT x = problem->data_slices[gpu][0].keys_in[t][i].GetSize();
                        printf("\t %lld", (long long) x);
                        double factor = 1.0 * x / problem->graph_slices[gpu]->in_counter[i];
                        if (factor > max_in_sizing_) max_in_sizing_ = factor;
                    }
            }
            if (num_gpus > 1) printf("\t %lld", (long long)(problem->data_slices[gpu]->frontier_queues[num_gpus].keys[0].GetSize()));
            printf("\n");
        }
        printf("\t queue_sizing =\t %lf \t %lf", max_queue_sizing_[0], max_queue_sizing_[1]);
        if (num_gpus > 1) printf("\t in_sizing =\t %lf", max_in_sizing_);
        printf("\n");
    }

    // Clean up
    if (org_size   ) { delete   org_size   ; org_size    = NULL; }
    if (problem    ) { delete   problem    ; problem     = NULL; }
    if (enactor    ) { delete   enactor    ; enactor     = NULL; }
    if (ref_rank   ) { delete[] ref_rank   ; ref_rank    = NULL; }
    if (ref_node_id) { delete[] ref_node_id; ref_node_id = NULL; }
    if (h_rank     ) { delete[] h_rank     ; h_rank      = NULL; }
    if (h_node_id  ) { delete[] h_node_id  ; h_node_id   = NULL; }
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 * @tparam DEBUG
 *
 * @param[in] info Pointer to mObject info.
 */
template <
    typename      VertexId,
    typename      Value,
    typename      SizeT,
    bool          INSTRUMENT,
    bool          DEBUG >
void RunTests_size_check(Info<VertexId, Value, SizeT> *info)
{
    if (info->info["size_check"].get_bool())
    {
        RunTests<VertexId, Value, SizeT, INSTRUMENT, DEBUG,  true>(info);
    }
    else
    {
        RunTests<VertexId, Value, SizeT, INSTRUMENT, DEBUG, false>(info);
    }
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 *
 * @param[in] info Pointer to mObject info.
 */
template <
    typename    VertexId,
    typename    Value,
    typename    SizeT,
    bool        INSTRUMENT >
void RunTests_debug(Info<VertexId, Value, SizeT> *info)
{
    if (info->info["debug_mode"].get_bool())
    {
        RunTests_size_check<VertexId, Value, SizeT, INSTRUMENT,  true>(info);
    }
    else
    {
        RunTests_size_check<VertexId, Value, SizeT, INSTRUMENT, false>(info);
    }
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] info Pointer to mObject info.
 */
template <
    typename      VertexId,
    typename      Value,
    typename      SizeT >
void RunTests_instrumented(Info<VertexId, Value, SizeT> *info)
{
    if (info->info["instrument"].get_bool())
    {
        RunTests_debug<VertexId, Value, SizeT,  true>(info);
    }
    else
    {
        RunTests_debug<VertexId, Value, SizeT, false>(info);
    }
}

/******************************************************************************
 * Main
 ******************************************************************************/

int main(int argc, char** argv)
{
    CommandLineArgs args(argc, argv);
    int graph_args = argc - args.ParsedArgc() - 1;
    if (argc < 2 || graph_args < 1 || args.CheckCmdLineFlag("help"))
    {
        Usage();
        return 1;
    }

    typedef int VertexId;  // Use int as the vertex identifier
    typedef float Value;   // Use float as the value type
    typedef int SizeT;     // Use int as the graph size type

    Csr<VertexId, Value, SizeT> csr(false);  // graph we process on
    Info<VertexId, Value, SizeT> *info = new Info<VertexId, Value, SizeT>;

    // graph construction or generation related parameters
    info->info["undirected"] = true;   // require undirected input graph
    info->info["edge_value"] = false;  // don't need per edge weight values

    info->Init("PageRank", args, csr);  // initialize Info structure
    RunTests_instrumented<VertexId, Value, SizeT>(info);  // run test

    return 0;
}

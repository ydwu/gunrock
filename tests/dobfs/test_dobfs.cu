// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_bfs.cu
 *
 * @brief Simple test driver program for breadth-first search.
 */

#include <stdio.h>
#include <string>
#include <deque>
#include <vector>
#include <iostream>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph construction utils
#include <gunrock/graphio/market.cuh>

// BFS includes
#include <gunrock/app/bfs/bfs_enactor.cuh>
#include <gunrock/app/bfs/bfs_problem.cuh>
#include <gunrock/app/bfs/bfs_functor.cuh>

// DOBFS includes
#include <gunrock/app/dobfs/dobfs_enactor.cuh>
#include <gunrock/app/dobfs/dobfs_problem.cuh>
#include <gunrock/app/dobfs/dobfs_functor.cuh>

// Operator includes
#include <gunrock/oprtr/advance/kernel.cuh>
#include <gunrock/oprtr/filter/kernel.cuh>

#include <moderngpu.cuh>

using namespace gunrock;
using namespace gunrock::app;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::bfs;
using namespace gunrock::app::dobfs;


/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/
void Usage()
{
    printf (
        " test_dobfs <graph type> <graph type args> [--device=<device_index>]\n"
        " [--src=<source_index>] [--instrumented] [--idempotence=<0|1>] [--v]\n"
        " [--undirected] [--iteration-num=<num>] [--quick=<0|1>] [--mark-pred]\n"
        " [--queue-sizing=<scale factor>]\n"
        "\n"
        "Graph types and args:\n"
        "  market <file>\n"
        "    Reads a Matrix-Market coordinate-formatted graph of directed / undirected\n"
        "    edges from stdin (or from the optionally-specified file).\n"
        "  --device=<device_index>   Set GPU device for running the test. [Default: 0].\n"
        "  --undirected              Treat the graph as undirected (symmetric).\n"
        "  --idempotence=<0 or 1>    Enable: 1, Disable: 0 [Default: Enable].\n"
        "  --instrumented            Keep kernels statics [Default: Disable].\n"
        "                            total_queued, search_depth and barrier duty\n"
        "                            (a relative indicator of load imbalance.)\n"
        "  --src=<source vertex id>  Begins BFS from the source [Default: 0].\n"
        "                            If randomize: from a random source vertex.\n"
        "                            If largestdegree: from largest degree vertex.\n"
        "  --quick=<0 or 1>          Skip the CPU validation: 1, or not: 0 [Default: 1].\n"
        "  --mark-pred               Keep both label info and predecessor info.\n"
        "  --queue-sizing=<factor>   Allocates a frontier queue sized at: \n"
        "                            (graph-edges * <scale factor>). [Default: 1.0]\n"
        "  --v                       Print verbose per iteration debug info.\n"
        "  --iteration-num=<number>  Number of runs to perform the test [Default: 1].\n"
    );
}

/**
 * @brief Displays the BFS result (i.e., distance from source)
 *
 * @param[in] source_path Search depth from the source for each node.
 * @param[in] preds Predecessor node id for each node.
 * @param[in] nodes Number of nodes in the graph.
 * @param[in] MARK_PREDECESSORS Whether to show predecessor of each node.
 * @param[in] ENABLE_IDEMPOTENCE Whether to enable idempotence mode.
 */
template<typename VertexId, typename SizeT>
void DisplaySolution(
    VertexId *source_path,
    VertexId *preds,
    SizeT nodes,
    bool MARK_PREDECESSORS,
    bool ENABLE_IDEMPOTENCE)
{
    if (nodes > 40) nodes = 40;
    printf("\nFirst %d labels of the GPU result.\n", nodes);

    printf("[");
    for (VertexId i = 0; i < nodes; ++i)
    {
        PrintValue(i);
        printf(":");
        PrintValue(source_path[i]);
        if (MARK_PREDECESSORS && !ENABLE_IDEMPOTENCE)
        {
            printf(",");
            PrintValue(preds[i]);
        }
        printf(" ");
    }
    printf("]\n");
}

/******************************************************************************
 * BFS Testing Routines
 *****************************************************************************/

/**
 * @brief A simple CPU-based reference BFS ranking implementation.
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] source_path Host-side vector to store CPU computed labels for each node
 * @param[in] src Source node where BFS starts
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT >
void SimpleReferenceBfs(
    const Csr<VertexId, Value, SizeT>       &graph,
    VertexId                                *source_path,
    VertexId                                src,
    bool                                    enable_idempotence,
    bool                                    quiet = false)
{
    // Initialize distances
    if (enable_idempotence)
    {
        for (VertexId i = 0; i < graph.nodes; ++i)
        {
            source_path[i] = -1;
        }
    }
    else
    {
        for (VertexId i = 0; i < graph.nodes; ++i)
        {
            source_path[i] = util::MaxValue<VertexId>() - 1;
        }
    }
    source_path[src] = 0;
    VertexId search_depth = 0;

    // Initialize queue for managing previously-discovered nodes
    std::deque<VertexId> frontier;
    frontier.push_back(src);

    // Perform BFS
    CpuTimer cpu_timer;
    cpu_timer.Start();
    while (!frontier.empty())
    {

        // Dequeue node from frontier
        VertexId dequeued_node = frontier.front();
        frontier.pop_front();
        VertexId neighbor_dist = source_path[dequeued_node] + 1;

        // Locate adjacency list
        int edges_begin = graph.row_offsets[dequeued_node];
        int edges_end = graph.row_offsets[dequeued_node + 1];

        for (int edge = edges_begin; edge < edges_end; ++edge)
        {
            //Lookup neighbor and enqueue if undiscovered
            VertexId neighbor = graph.column_indices[edge];
            if (source_path[neighbor] == -1 ||
                    source_path[neighbor] == util::MaxValue<VertexId>() - 1)
            {
                source_path[neighbor] = neighbor_dist;
                if (search_depth < neighbor_dist)
                {
                    search_depth = neighbor_dist;
                }
                frontier.push_back(neighbor);
            }
        }
    }

    cpu_timer.Stop();
    float elapsed = cpu_timer.ElapsedMillis();
    search_depth++;

    if (!quiet)
    {
        printf("CPU BFS finished in %lf msec. Search depth is: %d\n",
               elapsed, search_depth);
    }
}

/**
 * @brief Run BFS tests
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 * @tparam DEBUG
 * @tparam SIZE_CHECK
 * @tparam MARK_PREDECESSORS
 * @tparam ENABLE_IDEMPOTENCE
 *
 * @param[in] info Pointer to mObject info.
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT,
    bool DEBUG,
    bool SIZE_CHECK,
    bool MARK_PREDECESSORS,
    bool ENABLE_IDEMPOTENCE >
void RunTests(Info<VertexId, Value, SizeT> *info)
{
    typedef DOBFSProblem < VertexId,
            SizeT,
            Value,
            MARK_PREDECESSORS,
            ENABLE_IDEMPOTENCE,
            (MARK_PREDECESSORS && ENABLE_IDEMPOTENCE) >
            Problem; // does not use double buffer

    Csr<VertexId, Value, SizeT> *csr = (Csr<VertexId, Value, SizeT>*)info->csr_ptr;
    Csr<VertexId, Value, SizeT> *csc = (Csr<VertexId, Value, SizeT>*)info->csc_ptr;
    VertexId src            = info->info["source_vertex"].get_int64();
    int max_grid_size       = info->info["max_grid_size"].get_int();
    int num_gpus            = info->info["num_gpus"].get_int();
    double max_queue_sizing = info->info["max_queue_sizing"].get_real();
    bool quiet_mode         = info->info["quiet_mode"].get_bool();
    bool quick_mode         = info->info["quick_mode"].get_bool();
    bool undirected         = info->info["undirected"].get_bool();
    bool stream_from_host   = info->info["stream_from_host"].get_bool();
    int iterations          = info->info["num_iteration"].get_int();
    double alpha            = info->info["alpha"].get_real();
    double beta             = info->info["beta"].get_real();

    json_spirit::mArray device_list = info->info["device_list"].get_array();
    int* gpu_idx = new int[num_gpus];
    for (int i = 0; i < num_gpus; i++) gpu_idx[i] = device_list[i].get_int();

    // TODO: remove after merge mgpu-cq
    ContextPtr *context = (ContextPtr*)info->context;

    // allocate host-side label array (for both reference and GPU results)
    VertexId *reference_labels = (VertexId*)malloc(sizeof(VertexId) * csr->nodes);
    VertexId *h_labels         = (VertexId*)malloc(sizeof(VertexId) * csr->nodes);
    VertexId *reference_check  = (quick_mode) ? NULL : reference_labels;
    VertexId *h_preds          = NULL;
    if (MARK_PREDECESSORS)
    {
        h_preds = (VertexId*)malloc(sizeof(VertexId) * csr->nodes);
    }

    // Allocate BFS enactor map
    DOBFSEnactor<Problem, INSTRUMENT, DEBUG, SIZE_CHECK> enactor(gpu_idx);

    // Allocate problem on GPU
    Problem *problem = new Problem;

    util::GRError(problem->Init(
                      stream_from_host,
                      undirected,
                      *csr,
                      *csc,
                      num_gpus,
                      alpha,
                      beta),
                  "Problem DOBFS Initialization Failed", __FILE__, __LINE__);

    // compute reference CPU BFS solution
    if (reference_check != NULL)
    {
        if (!quiet_mode) { printf(" computing reference value ...\n"); }
        SimpleReferenceBfs(
            *csr,
            reference_check,
            src,
            ENABLE_IDEMPOTENCE,
            quiet_mode);
        if (!quiet_mode) { printf("\n"); }
    }

    double elapsed = 0.0f;

    // Perform BFS
    GpuTimer gpu_timer;

    for (int iter = 0; iter < iterations; ++iter)
    {
        util::GRError(problem->Reset(
                          src, enactor.GetFrontierType(), max_queue_sizing),
                      "DOBFS Problem Data Reset Failed", __FILE__, __LINE__);
        gpu_timer.Start();
        util::GRError(enactor.template Enact<Problem>(
                          *context, problem, src, max_grid_size),
                      "DOBFS Problem Enact Failed", __FILE__, __LINE__);
        gpu_timer.Stop();
        elapsed += gpu_timer.ElapsedMillis();
    }
    elapsed /= iterations;

    // Copy out results
    util::GRError(problem->Extract(h_labels, h_preds),
                  "DOBFS Problem Data Extraction Failed", __FILE__, __LINE__);

    // Verify the result
    if (reference_check != NULL)
    {
        if (!MARK_PREDECESSORS)
        {
            if (!quiet_mode) { printf("Validity: "); }
            CompareResults(
                h_labels, reference_check, csr->nodes, true, quiet_mode);
        }
    }

    if (!quiet_mode)
    {
        DisplaySolution(h_labels, h_preds, csr->nodes,
                        MARK_PREDECESSORS, ENABLE_IDEMPOTENCE);
    }

    info->ComputeTraversalStats(  // compute running statistics
        enactor.enactor_stats.GetPointer(), elapsed, h_labels);

    if (!quiet_mode)
    {
        info->DisplayStats();  // display collected statistics
    }

    info->CollectInfo();  // collected all the info and put into JSON mObject

    // Clean up
    if (problem) delete problem;
    if (reference_labels) free(reference_labels);
    if (h_labels) free(h_labels);
    if (h_preds) free(h_preds);

    hipDeviceSynchronize();
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 * @tparam DEBUG
 * @tparam SIZE_CHECK
 * @tparam MARK_PREDECESSORS
 *
 * @param[in] info Pointer to mObject info.
 */
template <
    typename    VertexId,
    typename    Value,
    typename    SizeT,
    bool        INSTRUMENT,
    bool        DEBUG,
    bool        SIZE_CHECK,
    bool        MARK_PREDECESSORS >
void RunTests_enable_idempotence(Info<VertexId, Value, SizeT> *info)
{
    if (info->info["idempotent"].get_bool())
    {
        RunTests <VertexId, Value, SizeT, INSTRUMENT, DEBUG, SIZE_CHECK,
                 MARK_PREDECESSORS, true > (info);
    }
    else
    {
        RunTests <VertexId, Value, SizeT, INSTRUMENT, DEBUG, SIZE_CHECK,
                 MARK_PREDECESSORS, false> (info);
    }
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 * @tparam DEBUG
 * @tparam SIZE_CHECK
 *
 * @param[in] info Pointer to mObject info.
 */
template <
    typename    VertexId,
    typename    Value,
    typename    SizeT,
    bool        INSTRUMENT,
    bool        DEBUG,
    bool        SIZE_CHECK >
void RunTests_mark_predecessors(Info<VertexId, Value, SizeT> *info)
{
    if (info->info["mark_predecessors"].get_bool())
    {
        RunTests_enable_idempotence<VertexId, Value, SizeT, INSTRUMENT,
                                    DEBUG, SIZE_CHECK,  true> (info);
    }
    else
    {
        RunTests_enable_idempotence<VertexId, Value, SizeT, INSTRUMENT,
                                    DEBUG, SIZE_CHECK, false> (info);
    }
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 * @tparam DEBUG
 *
 * @param[in] info Pointer to mObject info.
 */
template <
    typename      VertexId,
    typename      Value,
    typename      SizeT,
    bool          INSTRUMENT,
    bool          DEBUG >
void RunTests_size_check(Info<VertexId, Value, SizeT> *info)
{
    if (info->info["size_check"].get_bool())
    {
        RunTests_mark_predecessors<VertexId, Value, SizeT, INSTRUMENT,
                                   DEBUG,  true>(info);
    }
    else
    {
        RunTests_mark_predecessors<VertexId, Value, SizeT, INSTRUMENT,
                                   DEBUG, false>(info);
    }
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 *
 * @param[in] info Pointer to mObject info.
 */
template <
    typename    VertexId,
    typename    Value,
    typename    SizeT,
    bool        INSTRUMENT >
void RunTests_debug(Info<VertexId, Value, SizeT> *info)
{
    if (info->info["debug_mode"].get_bool())
    {
        RunTests_size_check<VertexId, Value, SizeT, INSTRUMENT,  true>(info);
    }
    else
    {
        RunTests_size_check<VertexId, Value, SizeT, INSTRUMENT, false>(info);
    }
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] info Pointer to mObject info.
 */
template <
    typename      VertexId,
    typename      Value,
    typename      SizeT >
void RunTests_instrumented(Info<VertexId, Value, SizeT> *info)
{
    if (info->info["instrument"].get_bool())
    {
        RunTests_debug<VertexId, Value, SizeT, true>(info);
    }
    else
    {
        RunTests_debug<VertexId, Value, SizeT, false>(info);
    }
}

/******************************************************************************
* Main
******************************************************************************/

int main( int argc, char** argv)
{
    CommandLineArgs args(argc, argv);
    int graph_args = argc - args.ParsedArgc() - 1;
    if (argc < 2 || graph_args < 1 || args.CheckCmdLineFlag("help"))
    {
        Usage();
        return 1;
    }

    typedef int VertexId;  // Use int as the vertex identifier
    typedef int Value;     // Use int as the value type
    typedef int SizeT;     // Use int as the graph size type

    Csr<VertexId, Value, SizeT> csr(false);  // CSR graph we process on
    Csr<VertexId, Value, SizeT> csc(false);  // CSC graph we process on
    Info<VertexId, Value, SizeT> *info = new Info<VertexId, Value, SizeT>;

    // graph construction or generation related parameters
    info->info["undirected"] = args.CheckCmdLineFlag("undirected");
    info->info["edge_value"] = false;  // don't need per edge weight values

    info->Init("DOBFS", args, csr, csc);  // initialize Info structure
    RunTests_instrumented<VertexId, Value, SizeT>(info);  // run test

    return 0;
}
